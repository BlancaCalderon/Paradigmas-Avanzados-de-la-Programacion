#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>



//define numero de filas y columnas del tablero (CUIDADO CAMBIAR A COGER POR CONSOLA QUE FILAS SE QUIERE)
__constant__ int* FILAS;
__constant__ int* COLUMNAS;


//Funcion que muestra el tablero por consola
void mostrarTablero(int* tablero, int numFilas, int numColumnas, int dificultad)
{
    int N = numFilas;
    int M = numColumnas;
    if (numColumnas > numFilas) {
        N = numColumnas;
        M = numFilas;
    }
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
        {
           // printf("%d  ", tablero[i * N + j]);
            
            int num = tablero[i * N + j];
            if (num > dificultad)
            {
                if (7 <= num && num <= 13)
                {
                    printf("RC%d  ", num % 7);
                }
                else
                {
                    printf("%c  ", (char)num);
                }
            }
            else
            {
                printf("%d  ", num);
            }
            
        }
        printf("\n");
    }
    printf("\n");
}

//Kernel que elimina la fila y columna de la posicion pasada (Bomba)
__global__ void kernelBomba(int* dev_tablero, int numFila, int numCol, int pos_encontrar)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int N = numFila;
    if (numCol > numFila)
    {
        N = numCol;
    }
    int pos = ((col * N) + fila);

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna a borrar
    int filaBorrar = pos_encontrar / numCol;
    int colBorrar = pos_encontrar - filaBorrar * numCol;

    if (numFila > col && numCol > fila)
    {
        //si posici�n actual esta en la fila o columna que queremos borrar
        if (filaBorrar == filaActual || colBorrar == colActual && 0 <= filaActual <= numFila && 0 <= colActual <= numCol && (pos < numCol * numFila))
        {
            dev_tablero[pos] = -1; //Indicamos que se borra
        }
    }
    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial
}


//Kernel que elimina los elementos adyacentes a una posici�n (radio 4 elementos) (TNT)
__global__ void kernelTNT(int* dev_tablero, int numFila, int numCol, int pos_encontrar)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int N = numFila;
    if (numCol > numFila)
    {
        N = numCol;
    }
    int pos = ((col * N) + fila);

    if (numFila > col && numCol > fila)
    {
        //Calcula fila y columna de la posici�n actual
        int filaActual = pos / numCol;
        int colActual = pos - filaActual * numCol;

        //Calcula fila y columna a borrar teniendo en cuenta el rango
        int filaBorrar = pos_encontrar / numCol;
        int colBorrar = pos_encontrar - filaBorrar * numCol;

        int filaBorrarDer = filaBorrar + 4;
        int colBorrarAbajo = colBorrar + 4;
        int filaBorrarIzq = filaBorrar - 4;
        int colBorrarArriba = colBorrar - 4;

        //si posici�n actual es adyacente y esta dentro del rango que queremos borrar (4)
        if (filaBorrarIzq <= filaActual <= filaBorrarDer && colBorrarArriba <= colActual <= colBorrarAbajo && 0 <= filaActual <= numFila && 0 <= colActual <= numCol && pos < (numCol * numFila))
        {
            dev_tablero[pos] = -1; //Indicamos que se borra
        }
    }
    
    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial

}


//Kernel que elimina todos las posiciones del color indicado (ROMPECABEZAS)
__global__ void kernelRompeCabezas(int* dev_tablero, int numFila, int numCol, int color, int pos_encontrar)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int N = numFila;
    if (numCol > numFila)
    {
        N = numCol;
    }
    int pos = ((col * N) + fila);

    if (numFila > col && numCol > fila)
    {
        //Calcula fila y columna de la posici�n actual
        int filaActual = pos / numCol;
        int colActual = pos - filaActual * numCol;

        //si posici�n actual tiene el color indicado se elimina
        if (dev_tablero[pos] == color && pos < (numCol * numFila))
        {

            dev_tablero[pos] = -1; //Indicamos que se borra
        }
    }

    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial

}


//Kernel que lleva a cabo la generacion del tablero de forma aleatoria
__global__ void kernelGenerarTablero(int* dev_tablero, int dev_semilla, int dificultad, int numCol, int numFila)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos

    int pos = ((col * numFila) + fila);

    if (numFila > col && numCol > fila)
    {
    //    printf("He entrado %d \n", pos);
        hiprandState_t state;
        hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
        dev_tablero[pos] = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
    }
}

__global__ void kernelReemplazarPosiciones(int* dev_tablero, int numFila, int numCol, int dev_semilla, int dificultad, int* dev_index)
{

    dev_index[0] = 0;                                       //Lo utilizamos para contabilizar el numero de llamadas que hay que realizar al kernel 
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int N = numFila;

    if (numCol > numFila)
    {
        N = numCol;
    }
    int pos = ((col * N) + fila);

    if (dev_tablero[pos] == -1 && numFila > col && numCol > fila)
    {

        int filaActual = pos / numCol;
        int colActual = pos - filaActual * numCol;
        if (filaActual > 0 && filaActual <= numFila && dev_tablero[pos - numCol] != -1)
        {
            dev_tablero[pos] = dev_tablero[pos - numCol];
            dev_tablero[pos - numCol] = -1;
            atomicAdd(&dev_index[0], 1);
        }
        else if (dev_tablero[pos - numCol] != -1)
        {
            hiprandState_t state;
            hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
            int color = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
            printf("COLOR %d\n", color);
            dev_tablero[pos] = color;
            atomicAdd(&dev_index[0], 1);
        }
    }

}

__global__ void kernelEncontrarCaminos(int* dev_tablero, int numFila, int numCol, int* dev_index, int pos_encontrar, bool* dev_encontrado, int color)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int N = numFila;

    if (numCol > numFila)
    {
        N = numCol;
    }

    int pos = ((col * N) + fila);
    bool encontrado = false;
    bool camino_invalido = false;
    int posAux;
    int index = 0;

    //Recorrer 1� fila y 2�Columna del tablero en la que se encuentra la celda de POS
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;
    int ultima_posicion = pos;
    __shared__ int* tablero_compartido;

    if ((dev_tablero[pos] == color || dev_tablero[pos] == -1) && pos_encontrar == pos && (numFila > col) && (numCol > fila))
    {
        printf("Hilo %d ha entrado a buscar camino [%d][%d]\n", pos, col, fila);
        encontrado = false;
        posAux = pos;

        while ((posAux < (numCol*numFila)) && !camino_invalido && !encontrado)// && (dev_tablero[posAux] == color || dev_tablero[posAux] == -1))
        {
            int sigfila = (posAux + 1) / numCol;                 //Fila en la que se encuentra el siguiente elemento
            int sigcol = (posAux + 1) - sigfila * numCol;       //Columna en la que se encuentra el siguiente elemento

            int fila_anterior = (posAux - 1) / numCol;                 //Fila en la que se encuentra el elemento ANTERIOR
            int col_anterior = (posAux - 1) - fila_anterior * numCol; //Columna en la que se encuentra el elemento anterior

            int posSigFila = (posAux + numCol) / numCol;
            int fila_actual = posAux / numCol;
            int col_actual = posAux - fila_actual * numCol;
            
            printf("\n*Condicion ABAJO lleva a la posicion[%d] desde pos[%d] hilo %d con fila actual %d con color %d\n", posAux + numCol, posAux, pos, fila_actual, dev_tablero[posAux+numCol]);

            printf("\n*Condicion IZQUIERDA lleva a la posicion[%d] desde pos[%d] hilo %d con columna anterior %d\n", posAux - 1, posAux, pos, col_anterior);

            printf("\n*Condicion ARRIBA lleva a la posicion[%d] desde pos[%d] hilo %d con color %d\n", posAux - numCol, posAux, pos, color);

            if (color == dev_tablero[posAux + 1]  && sigcol >= 0 && (posAux + 1) != ultima_posicion)          //Nos desplazamos a la derecha
            {
                printf("\nCondicion DERECHA lleva a la posicion[%d] desde pos[%d] hilo %d con color %d\n", posAux + 1, pos, posAux, color);
                printf("\nAvanza a la pos DERECHA [%d] hilo %d con color %d\n", posAux, pos, color);
                index += 1;
                ultima_posicion = posAux;
                posAux += 1;
                dev_tablero[posAux] = -1;

            }
            else if (color == dev_tablero[posAux + numCol] && (posAux + numCol) < (numCol*numFila))  //Hacia abajo  && (posAux + numCol) != ultima_posicion
            {
                ultima_posicion = posAux;
                posAux = posAux + numCol;
                index += 1;
                dev_tablero[posAux] = -1;
                printf("\nAvanza a la pos de ABAJO [%d] ultima posicion %d hilo %d", posAux + numCol, posAux, pos);
            }
            else if (color == dev_tablero[posAux - 1] && col_anterior >= 0 && (posAux - 1) != ultima_posicion)           //Izquierda
            {
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux - 1;
                printf("\nAvanza a la pos IZQUIERDA [%d] hilo %d", posAux, pos);

                dev_tablero[posAux] = -1;
            }
            else if (color == dev_tablero[posAux - numCol] && (posAux - numCol) >= 0 && filaActual >= 0 && filaActual <= numFila && (posAux - numCol) != ultima_posicion)  //ARRIBA
            {
                
                index += 1;
                ultima_posicion = posAux;
                printf("\nAvanza a la pos ARRIBA [%d] ultima posicion %d hilo %d", (posAux - numCol), ultima_posicion, pos);
                posAux = posAux - numCol;
                dev_tablero[posAux] = -1;
            }
            else
            {
                printf("\nNumero elementos %d\n", dev_index[0]);

                printf("\nCamino ENCONTRADO [%d]\n", pos);

                if (index > 0) {
                    atomicAdd(&dev_index[0], 1);
                    encontrado = true;
                }
                else {
                    encontrado = false;
                }
                
                printf("\nCamino no encontrado desde la posicion %d index vale %d\n", posAux, index);
                camino_invalido = true;
            }

        }
        dev_encontrado[0] = encontrado;
        printf("DEV_ENCONTRADO %d \n", dev_encontrado[0]);
        printf("DEV_INDEX %d \n", dev_index[0]);
        if (dev_index[0] >= 1 && pos == pos_encontrar)
        {
            printf("Posicion a encontrar %d \n", pos_encontrar);
            dev_tablero[pos_encontrar] = -1;              //Establecemos la posicion a encontrar en -1
        }
    }
    __syncthreads();

}

__global__ void kernelEncontrarBomba(int* dev_tablero, int numFila, int numCol, int pos_encontrar, int* dev_index_fila, int* dev_index_col)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int N = numFila;

    if (numCol > numFila)
    {
        N = numCol;
    }

    int pos = ((col * N) + fila);

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna de la posici�n a encontrar
    int filaEncontrar = pos_encontrar / numCol;
    int colEncontrar = pos_encontrar - filaEncontrar * numCol;

    if (filaActual == filaEncontrar && (int)dev_index_fila > 5 && dev_tablero[pos] == -1 && (numFila > col) && (numCol > fila))
    {
        atomicAdd(&dev_index_fila[0], 1);
    }

    if (colActual == colEncontrar && (int)dev_index_col > 5 && dev_tablero[pos] == -1 && (numFila > col) && (numCol > fila))
    {
        atomicAdd(&dev_index_col[0], 1);
    }

    __syncthreads();
    if (dev_index_fila[0] != dev_index_col[0] && (numFila > col) && (numCol > fila))
    {
      //  printf("Valor del contador de fila %d y contador columna %d \n", dev_index_col[0], dev_index_fila[0]);
        if ((dev_index_fila[0] == 5 && dev_index_col[0] == 1) || (dev_index_col[0] == 5 && dev_index_fila[0] == 1))
        {
            dev_tablero[pos_encontrar] = 'B';
        }
    }
}

__global__ void kernelEncontrarRompecabezas(int* dev_tablero, int numFila, int numCol, int pos_encontrar, int* dev_index, int dev_semilla, int dificultad)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int N = numFila;

    if (numCol > numFila)
    {
        N = numCol;
    }

    int pos = ((col * N) + fila);

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna de la posici�n a encontrar
    int filaEncontrar = pos_encontrar / numCol;
    int colEncontrar = pos_encontrar - filaEncontrar * numCol;

    if (dev_tablero[pos] == -1 && (int)dev_index > 7 && (numFila > col) && (numCol > fila))
    {
        atomicAdd(&dev_index[0], 1);
    }
    __syncthreads();

    if (dev_index[0] >= 7 && pos == pos_encontrar && (numFila > col) && (numCol > fila))
    {
      //  printf("Valor del contador de rompecabezas %d \n", dev_index[0]);
        hiprandState_t state;
        hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
        int color = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
     //   printf("Soy el hilo %d voy a actualizar el tablero \n ", pos);
        int colorS = 7 + color;
        dev_tablero[pos_encontrar] = colorS;
    }

}


//Inicializamos el tablero
int* inicializarTablero(int* h_tablero, int size, int numCol, int numFila, int dificultad, int hilosBloqueX, int hilosBloqueY, int gridX, int gridY)
{
    int* (dev_Tablero);

    //Reservar espacio en memoria para GPU (2 matrices y matriz resultado)
    hipMalloc((void**)&dev_Tablero, size * sizeof(int));

    //Copiamos datos a la GPU 
    hipMemcpy(dev_Tablero, h_tablero, size * sizeof(int), hipMemcpyHostToDevice);

    unsigned int semilla = time(NULL);
    dim3 dimGrid(gridX, gridY);
    dim3 dimBlock(hilosBloqueX, hilosBloqueY);
    kernelGenerarTablero << <dimGrid, dimBlock >> > (dev_Tablero, semilla, dificultad, numCol, numFila);

    // Copiamos de la GPU a la CPU
    hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);

    return h_tablero;

}

//Funcion que llama a kernel para encontrar todos los caminos hacia bloque indicado
int encontrarCamino(int* h_tablero_original, int numFilas, int numColumnas, int coordX, int coordY, int dificultad, int vida, int hilosBloqueX, int hilosBloqueY, int gridX, int gridY)
{
    int* h_tablero = h_tablero_original;
    int* (dev_Tablero), * (dev_index), * (dev_index_fila), * (dev_index_col), * (dev_index_RC);
    bool* dev_encontrado;
    int size = numFilas * numColumnas;
    bool h_encontrado = true;
    int* h_index = { 0 };
    int* h_index_col = { 0 };
    int* h_index_fila = { 0 };
    int* h_index_RC = { 0 };

    int pos_encontrar = coordX * numFilas + coordY;   //Posicion a ENCONTRAR en el vector 1D
    if (numColumnas > numFilas) 
    {
        pos_encontrar = coordX * numColumnas + coordY;   //Posicion a ENCONTRAR en el vector 1D   
    }
    
    int color = h_tablero[pos_encontrar];

    unsigned int semilla = time(NULL);
    printf("Posicion a ENCONTRAR %d\n", pos_encontrar);
    //Reservar espacio en memoria para GPU (2 matrices y matriz resultado)
    hipMalloc((void**)&dev_Tablero, size * sizeof(int));
    hipMalloc((void**)&dev_index, sizeof(int));
    hipMalloc((void**)&dev_index_col, sizeof(int));
    hipMalloc((void**)&dev_index_fila, sizeof(int));
    hipMalloc((void**)&dev_index_RC, sizeof(int));
    hipMalloc(&dev_encontrado, sizeof(bool));

    //Copiamos datos a la GPU 
    hipMemcpy(dev_Tablero, h_tablero, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index, h_index, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_col, h_index_col, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_fila, h_index_fila, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_RC, h_index_RC, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_encontrado, &h_encontrado, sizeof(bool), hipMemcpyHostToDevice);

    dim3 dimGrid(gridX, gridY);
    dim3 dimBlock(hilosBloqueX, hilosBloqueY);

    //Segun si es alguno de los bloques especiales o es una jugada normal (66 --> B, 84 --> T,)
    int contenido = h_tablero[pos_encontrar];

    if (contenido == 'B')
    {
        kernelBomba << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }

    else if (contenido == 'T')
    {
        kernelTNT << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }

    else if (7 <= contenido && contenido <= 13) //Si es RC
    {
        int colorBorrar = contenido % 7;
        kernelRompeCabezas << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, colorBorrar, pos_encontrar);
        printf("COLOrrr %d \n", colorBorrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }
    else //Si es bloque simple
    {
        int cont = 0;
        //Desde posici�n idicada se encuentran todos los caminos con el mismo color
        while (h_encontrado)
        {
            printf("contador %d \n", cont);
            kernelEncontrarCaminos << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, dev_index, pos_encontrar, dev_encontrado, color);
            hipMemcpy(&h_encontrado, dev_encontrado, sizeof(bool), hipMemcpyDeviceToHost);
            hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(&h_index, dev_index, sizeof(int), hipMemcpyDeviceToHost);
            printf("Valor del puntero %d \n", h_encontrado);
            printf("H_inxex %d\n", h_index);
            cont += 1;

            mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
        }
        if ((int)h_index == 0 && vida >= 1)
        {
            vida = vida - 1;
        }
        h_index_fila = { 0 };
        h_index_col = { 0 };
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
        kernelEncontrarBomba << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar, dev_index_fila, dev_index_col);
        hipMemcpy(&h_index_fila, dev_index_fila, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_index_col, dev_index_col, sizeof(int), hipMemcpyDeviceToHost);

        printf("N Filas %d - N Columnas %d \n", h_index_fila, h_index_col);
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);

        kernelEncontrarRompecabezasTNT << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar, dev_index_RC, semilla, dificultad);
        hipMemcpy(&h_index_RC, dev_index_RC, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);

    }
    mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);


    h_index = { 0 };
    int iteraciones = 10;
    //Bucle para reemplazar las posiciones eliminadas mientras que se pueda hacer algun cambio y si no termine
    while (iteraciones > 0)
    {
        kernelReemplazarPosiciones << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, semilla, dificultad, dev_index);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_index, dev_index, sizeof(int), hipMemcpyDeviceToHost);
        iteraciones = (int)h_index;
    }
    mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);

    hipFree(dev_encontrado);
    hipFree(dev_Tablero);
    hipFree(dev_index);
    hipFree(dev_index_fila);
    hipFree(dev_index_col);

    return vida;
}

void main(int argc, char* argv[])
{
    //Declaracion variables
    //int* h_tablero;
    int numFilas = 3;
    int numColumnas = 9;
    int coordenadaX;
    int coordenadaY;
    int size = numFilas * numColumnas;
    int dificultad = 4;
    bool terminado = false;
    int vida = 5;
    char modoJuego = 'A';
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    char* nombre = deviceProp.name;
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    int maxThreadsSM = deviceProp.maxThreadsPerMultiProcessor;
    int maxBlockx = deviceProp.maxGridSize[0];
    int maxBlocky = deviceProp.maxGridSize[1];
    int maxGridX = deviceProp.maxThreadsDim[0];
    int maxGridY = deviceProp.maxThreadsDim[1];

    printf("Nombre del Device: %s\n", deviceProp.name);
    printf("Num maximo de hilos por bloque: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Num maximo de bloque s: %d\n", maxBlockx);
    printf("Dimensiones maximas para organizar los hilos en bloques (%d, %d, %d):\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("Dimensiones maximas para organizar los bloques en el grid (%d, %d, %d):\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

    int hilosBloqueX = ceil(numFilas / (float)2);
    int hilosBloqueY = ceil(numColumnas / (float)2);
    int gridX = ceil(numFilas / (float)hilosBloqueX);
    int gridY = ceil(numColumnas / (float)hilosBloqueY);
    printf("dimBlock(%d, %d), dimGrid(%d, %d): ", hilosBloqueX, hilosBloqueY, gridX, gridY);

    //Pasamos a memoria constante el numero de filas y columnas introducidas por el usuario
    hipMemcpyToSymbol(HIP_SYMBOL(FILAS), &numFilas, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(COLUMNAS), &numColumnas, sizeof(int));

    //Reservamos memoria para el tablero, ya que no esta inicializado
    //int* h_tablero = (int*)malloc(numFilas * numColumnas * sizeof(int));

    //Llamamos a la funcion que inicializa con valores aleatorios el tablero
    //h_tablero = inicializarTablero(h_tablero, size, numColumnas, numFilas, dificultad, hilosBloqueX, hilosBloqueY, gridX, gridY);
    //int h_tablero[25] = { 3,3,3,3,4,3,3,4,3,1,4,3,'B',3,1,3,1,3,3,3,4,1,1,4,3};
    int h_tablero[27] = { 3,3,3,3,3,3,3,4,4,4,4,3,1,3,3,3,4,3,3,3,4,3,3,4,3,4,4 };
    // int h_tablero[25] = { 3,2,1,5,5,3,3,6,7,3,9,3,'B',3,1,3,1,3,3,3,4,1,1,4,3 };
     //Mostramos el tablero



     //Codigo para ejecutar programa y recibir datos por comando
     //Controla que no de error la llamada
    if (argc == 1)  //No se ha ejecutado por comando
    {
        printf("\nElija el modo de juego: A (Automatico) - M (Manual):  \n");
        scanf("%c", &modoJuego);
        printf("Modo de juego seleccionado: %c \n", modoJuego);
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
    }

    else if (argc == -1)
    {
        printf("ERROR en ejecucion\n");
    }
    //Controla que tengamos los argumentso encesarios (tipo de ejecucion, dificultad, filas, columnas)
    else if (argc < 5)
    {
        printf("ERROR: faltan argumentos de entrada\n");
        printf("%d \n", argc);
    }
    //Controlan que no se pasen mas argumentos de los deseados
    else if (argc > 5)
    {
        printf("ERROR: sobran argumentos de entrada\n");
    }
    else //Realiza llamada con los 4 argumemtos
    {
        //Guarda los argumentos pasadas en las respectivas variables
        modoJuego = (char)argv[1];
        dificultad = std::stoi(argv[2]);    //Guarda valor argumentos usando funcion stoi para convertirlo a int
        numFilas = std::stoi(argv[3]);
        numColumnas = std::stoi(argv[4]);

    }
    if (hilosBloqueX >= maxBlockx && hilosBloqueY >= maxBlocky && gridX > maxGridX && gridY > maxGridY)
    {
        printf("\nSe sobrepasan las dimensiones asociadas a la tarjeta grafica  \n");
    }
    else
    {
        while (vida > 0)
        {
            if (modoJuego == 'M' || modoJuego == 'm')
            {
                printf("\nIntroduzca las coordenadas del bloque que desea eliminar (x, y):  \n");
                scanf("%d %d", &coordenadaX, &coordenadaY);
            }
            else if (modoJuego == 'A' || modoJuego == 'a')
            {
                coordenadaX = (rand() % numFilas);
                coordenadaY = (rand() % numColumnas);
                printf("\nCoordenadas (%d, %d)  \n", coordenadaX, coordenadaY);
            }

            if ((coordenadaX < numFilas) && (coordenadaY < numColumnas) && (coordenadaX >= 0) && (coordenadaY >= 0))
            {
                vida = encontrarCamino(h_tablero, numFilas, numColumnas, coordenadaX, coordenadaY, dificultad, vida, hilosBloqueX, hilosBloqueY, gridX, gridY);
                printf("\nVida restante: %d \n", vida);
            }
            else
            {
                printf("\nLas coordenadas introducidas se encuentran fuera del rango del tablero [%d][%d] \n", numFilas, numColumnas);
            }


        }
        printf("\nPERDEDOR \n");
    }

}

