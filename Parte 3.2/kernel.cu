#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>



//define numero de filas y columnas del tablero (CUIDADO CAMBIAR A COGER POR CONSOLA QUE FILAS SE QUIERE)
__constant__ int* FILAS;
__constant__ int* COLUMNAS;
int vida = 5;
__constant__ int hiloX;
const int TESELAX = 5;
const int TESELAY = 2;

const int N_FILAS = 5;
const int M_COLUMNA = 2;

//Funcion que muestra el tablero por consola
void mostrarTablero(int* tablero, int numFilas, int numColumnas, int dificultad)
{
    int N = numFilas;
    int M = numColumnas;

    if (numFilas > numColumnas || numColumnas > numFilas) {
        N = numColumnas;
        M = numFilas;
    } 

    printf("Mostrar tablero - Valor de N = %d \n ", N);
    for (int i = 0; i < numFilas; i++)
    {
        for (int j = 0; j < numColumnas; j++)
        {
            // printf("%d  ", tablero[i * N + j]);

            int num = tablero[i * N + j];
            if (num > dificultad)
            {
                if (7 <= num && num <= 13)
                {
                    printf("RC%d  ", num % 7);
                }
                else
                {
                    printf("%c  ", (char)num);
                }
            }
            else
            {
                printf("%d  ", num);
            }

        }
        printf("\n");
    }
    printf("\n");
}


//Kernel que lleva a cabo la generacion del tablero de forma aleatoria
__global__ void kernelGenerarTablero(int* dev_tablero, int dev_semilla, int dificultad, int numCol, int numFila, int hilosBloqueX, int hilosBloqueY)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int N = numFila;
    int dim = blockDim.x;

    int pos = ((col * N) + fila);
    if (numCol > numFila)
    {
        N = numCol;
        dim = blockDim.y;
        pos = ((fila * N) + col);
    }
    int id = threadIdx.x * dim + threadIdx.y;
    id = threadIdx.y * dim + threadIdx.x;
    int tamX = numFila;

   // printf(" F[%d] C[%d] dim(%d, %d) | id = %d | pos = %d\n", fila, col,numFila, numCol, id, pos);

    __shared__ int t_compartido[TESELAX*TESELAY];

    int sig_bloqueX = (blockIdx.x + 1) % blockDim.x;
    int sig_bloqueY = (blockIdx.y + 1) % blockDim.y;

    int bloqueX = blockIdx.x * blockDim.x;
    int bloqueY = blockIdx.y * blockDim.y;

    if (numFila > fila && numCol > col)
    {
        printf("N� Bloque (%d, %d) hilo %d pos en memoria COMPARTIDA %d \n", blockIdx.x, blockIdx.y, pos, id);// blockIdx.x* dim + threadIdx.x, blockIdx.y* dim + threadIdx.y);
        hiprandState_t state;
        hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
        t_compartido[id] = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
        printf("Valor en memoria COMPARTIDA %d\n", t_compartido[id]);
        __syncthreads();
        dev_tablero[pos] = t_compartido[id];
    }

}
//Kernel que elimina la fila y columna de la posicion pasada (Bomba)
__global__ void kernelBomba(int* dev_tablero, int numFila, int numCol, int pos_encontrar)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    
    int sig_bloqueX = (blockIdx.x + 1) * blockDim.x;
    int sig_bloqueY = (blockIdx.y + 1) * blockDim.y;    //POSICION INICIAL del siguiente hilo en el siguiente bloque
    
    int bloqueX = blockIdx.x * blockDim.x;
    int bloqueY = blockIdx.y * blockDim.y;
    
    int N = numFila;
    int dim = blockDim.x;

    int pos = ((col * N) + fila);
    if (numCol >= numFila)
    {
        N = numCol;
        dim = blockDim.y;
        pos = ((fila * N) + col);
        //id = blockIdx.y * dim + threadIdx.y;
    }
    int id = threadIdx.x * dim + threadIdx.y;
    id = threadIdx.y * dim + threadIdx.x;
    int tamX = numFila;

    

    __shared__ int t_compartido[TESELAX*TESELAY];

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna a borrar
    int filaBorrar = pos_encontrar / numCol;
    int colBorrar = pos_encontrar - filaBorrar * numCol;
    int* contenido = dev_tablero;

  //  printf("Dim Col %d - Dim Fila %d - Id[%d]  Pos[%d]\n", blockDim.x, blockDim.y, id, pos);

    if (numFila > fila && numCol > col)
    {
        t_compartido[id] = dev_tablero[pos];
        __syncthreads();
        //si posici�n actual esta en la fila o columna que queremos borrar
        if (filaBorrar == filaActual || colBorrar == colActual) //&& 0 <= filaActual <= numFila && 0 <= colActual <= numCol)
        {
           // printf(" F[%d] C[%d] dim(%d, %d) | id = %d | pos = %d\n", fila, col, numFila, numCol, id, pos);
            if (col < sig_bloqueX && fila < sig_bloqueY && col >= bloqueX  && fila >= bloqueY) 
            {
            //    printf("H%d | Bloque Actual  [%d][%d] || Sig Bloques [%d][%d]\n",pos, bloqueX, bloqueY, sig_bloqueX, sig_bloqueY);
                t_compartido[id] = -1;  //Indicamos que se borra
                dev_tablero[pos] = t_compartido[id];
            }
            else {
                dev_tablero[pos] = -1;
            }
            
            
        }
    }

    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial
    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    if (numFila > fila && numCol > col)
    {
        dev_tablero[pos] = t_compartido[id];  //Indicamos que se borra
    }
    
   
}


//Kernel que elimina los elementos adyacentes a una posici�n (radio 4 elementos) (TNT)
__global__ void kernelTNT(int* dev_tablero, int numFila, int numCol, int pos_encontrar)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int sig_bloqueX = (blockIdx.x + 1) * blockDim.x;
    int sig_bloqueY = (blockIdx.y + 1) * blockDim.y;
    int bloqueX = blockIdx.x * blockDim.x;
    int bloqueY = blockIdx.y * blockDim.y;
    int N = numFila;
    int dim = blockDim.x;

    int pos = ((col * N) + fila);
    if (numCol > numFila)
    {
        N = numCol;
        dim = blockDim.y;
        pos = ((fila * N) + col);
        //id = blockIdx.y * dim + threadIdx.y;
    }
    int id = threadIdx.x * dim + threadIdx.y;
    id = threadIdx.y * dim + threadIdx.x;
    int tamX = numFila;

  //  printf(" F[%d] C[%d] dim(%d, %d) | id = %d | pos = %d\n", fila, col, numFila, numCol, id, pos);

    __shared__ int t_compartido[TESELAX * TESELAY];

    if (numFila > fila && numCol > col)
    {
        //Calcula fila y columna de la posici�n actual
        int filaActual = pos / numCol;
        int colActual = pos - filaActual * numCol;

        //Calcula fila y columna a borrar teniendo en cuenta el rango
        int filaBorrar = pos_encontrar / numCol;
        int colBorrar = pos_encontrar - filaBorrar * numCol;

        int filaBorrarDer = filaBorrar + 4;
        int colBorrarAbajo = colBorrar + 4;
        int filaBorrarIzq = filaBorrar - 4;
        int colBorrarArriba = colBorrar - 4;

        //si posici�n actual es adyacente y esta dentro del rango que queremos borrar (4)
        if (filaBorrarIzq <= filaActual <= filaBorrarDer && colBorrarArriba <= colActual <= colBorrarAbajo && 0 <= filaActual <= numFila && 0 <= colActual <= numCol && pos < (numCol * numFila))
        {
            dev_tablero[pos] = -1; //Indicamos que se borra
        }
    }

    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial

}


//Kernel que elimina todos las posiciones del color indicado (ROMPECABEZAS)
__global__ void kernelRompeCabezas(int* dev_tablero, int numFila, int numCol, int color, int pos_encontrar)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int sig_bloqueX = (blockIdx.x + 1) * blockDim.x;
    int sig_bloqueY = (blockIdx.y + 1) * blockDim.y;
    int bloqueX = blockIdx.x * blockDim.x;
    int bloqueY = blockIdx.y * blockDim.y;
    int N = numFila;
    int dim = blockDim.x;

    int pos = ((col * N) + fila);
    if (numCol > numFila)
    {
        N = numCol;
        dim = blockDim.y;
        pos = ((fila * N) + col);
    }
    int id = threadIdx.x * dim + threadIdx.y;
    id = threadIdx.y * dim + threadIdx.x;
    int tamX = numFila;

  //  printf(" F[%d] C[%d] dim(%d, %d) | id = %d | pos = %d\n", fila, col, numFila, numCol, id, pos);

    __shared__ int t_compartido[TESELAX * TESELAY];

    if (numFila > fila && numCol > col)
    {
        //Calcula fila y columna de la posici�n actual
        int filaActual = pos / numCol;
        int colActual = pos - filaActual * numCol;

        //si posici�n actual tiene el color indicado se elimina
        if (dev_tablero[pos] == color && pos < (numCol * numFila))
        {

            dev_tablero[pos] = -1; //Indicamos que se borra
        }
    }

    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial

}

/*
__global__ void kernelPrueba(int* dev_tablero, int numFila, int numCol)
{

    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos

    int sig_bloqueX = (blockIdx.x + 1) * blockDim.x;
    int sig_bloqueY = (blockIdx.y + 1) * blockDim.y;    //POSICION INICIAL del siguiente hilo en el siguiente bloque
    int bloque_col_anteriorX = (blockIdx.x - 1) * blockDim.x;
    int bloque_fila_anteriorY = (blockIdx.y - 1) * blockDim.y;

    int bloqueX = blockIdx.x * blockDim.x;
    int bloqueY = blockIdx.y * blockDim.y;

    int bloqueX_anterior = (blockIdx.x) * blockDim.x;
    int bloqueY_siguiente = blockIdx.y * blockDim.y;

    int N = numFila;
    int dim = blockDim.x;

    int pos = ((col * N) + fila);
    int id_general = bloqueX * N + bloqueY;
    // int id_fila_anterior = (bloqueX)*N + bloqueY;
    int id_fila_anterior = ((fila - 1) * gridDim.x) + col;
   
    if (numCol > numFila)
    {
        N = numCol;
        dim = blockDim.y;
        pos = ((fila * N) + col);
        id_general = bloqueY * N + bloqueX;
        id_fila_anterior = (bloqueY - 1) * N + (bloqueX);

    }
    
    int posAnterior = (pos - numCol) / dim;
    printf("Hilo [%d] - F[%d]C[%d]\n", pos, fila, col);
    __syncthreads();
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int blockId_fila = (blockIdx.x - 1) + (blockIdx.y) * gridDim.x;

    int tileRow = blockId / (N / 2);
    int tileCol = (blockId % (N / 2)) * 5;

    int bloque_anterior = blockIdx.x - ((threadIdx.x + blockDim.x * threadIdx.y) == 0 ? 1 : 0);
    int fila_anterior = fila - 1;
    int indice_fila_anterior = fila_anterior * N + col;
    int indice_bloque_anterior = bloque_anterior * blockDim.x * blockDim.y;
    int previous_index = indice_bloque_anterior + indice_fila_anterior;

    int tile_size_x = blockDim.x * gridDim.x; // ancho total de la tesela
    int tile_size_y = blockDim.y * gridDim.y; // alto total de la tesela
    int tile_x = blockIdx.x * blockDim.x; // posici�n x de la esquina superior izquierda de la tesela
    int tile_y = blockIdx.y * blockDim.y; // posici�n y de la esquina superior izquierda de la teselab
    int pos_x = threadIdx.x + tile_x; // posici�n x absoluta del hilo en la matriz
    int pos_y = threadIdx.y + tile_y; // posici�n y absoluta del hilo en la matriz

    int tile_idx_x = pos_x / tile_size_x; // �ndice x de la tesela a la que pertenece la posici�n
    int tile_idx_y = pos_y / tile_size_y; // �ndice y de la tesela a la que pertenece la posici�n


    int bloque_arriba_y = (fila - 1) / TESELAY;
    int bloque_abajo_y = (fila + 1) / TESELAY;
    int bloque_izquierda_col = (col - 1) / TESELAX;
    int bloque_derecha_col = (col + 1) / TESELAX;

    //Id bloque actual del hilo en X e Y
    int bloque_fila_actual = (fila) / TESELAY;
    int bloque_col_actual = (col) / TESELAX;
    

    printf("Hilo [%d] - Id Bloques: Fila Actual[%d] Arriba[%d], Col actual[%d]Izquierda[%d]\n", pos, bloque_fila_actual, bloque_arriba_y, bloque_col_actual, bloque_izquierda_col);

}
*/
__global__ void kernelReemplazarPosiciones(int* dev_tablero, int numFila, int numCol, int dev_semilla, int dificultad, int* dev_index)
{
    dev_index[0] = 0;
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos

    int sig_bloqueX = (blockIdx.x + 1) * blockDim.x;            //Posicion inicial de la columna del siguiente hilo del siguiente bloque
    int sig_bloqueY = (blockIdx.y + 1) * blockDim.y;            //Posicion inicial de la fila del siguiente hilo en el siguiente bloque

    int bloqueX = blockIdx.x * blockDim.x;                      // //Posicion inicial de la columna del primer hilo del blqoue actual
    int bloqueY = blockIdx.y * blockDim.y;                      //Posicion inicial de la fila del primer hilo del blqoue actual
   
    int N = numFila;
    int dim = blockDim.x;
    int pos = ((col * N) + fila);
    int M = numCol;


    int bloque_fila_actual = (fila) / TESELAX;
    int bloque_col_actual = (col) / TESELAY;

    //Calculo del id del bloque tiene el hilo en la posicion de su izquierda, derecha, arriba y abajo
    int bloque_arriba_y = (fila - 1) / TESELAY;         //Arriba -1 Fila
    int bloque_abajo_y = (fila + 1) / TESELAY;         //Derecha +1
    int bloque_izquierda_col = (col - 1) / TESELAX;     //Abajo +1 Fila
    int bloque_derecha_col = (col + 1) / TESELAX;       //Izquierda -1

   // printf("Hilo [%d] -divido %d / %d\n", pos, fila-1, TESELAY);
    
    if (numCol >= numFila)
    {
        N = numCol;
        M = numFila;
        dim = blockDim.y;
        pos = ((fila * N) + col);

        //Id bloque actual del hilo en X e Y
        bloque_fila_actual = (fila) / TESELAY;
        bloque_col_actual = (col) / TESELAX;
       
        bloque_arriba_y = (fila - 1) / TESELAY;         //Arriba -1 Fila
        bloque_abajo_y = (fila + 1) / TESELAY;         //Derecha +1
        bloque_izquierda_col = (col - 1) / TESELAX;     //Abajo +1 Fila
        bloque_derecha_col = (col + 1) / TESELAX;       //Izquierda -1

    }

    int id = threadIdx.y * dim + threadIdx.x;                   //Id del hilo dentro de la tesela

   // printf("Hilo [%d] - F[%d]C[%d]\n", pos, fila, col);
   // printf("Hilo [%d] - B (col, fila)=(%d, %d) Arriba[%d]\n", pos, bloque_col_actual, bloque_fila_actual, bloque_arriba_y);

//    printf("Hilo [%d] - Id Bloques: Fila Actual[%d] Arriba[%d], Col actual[%d]Izquierda[%d]\n", pos, bloque_fila_actual, bloque_arriba_y,bloque_col_actual, bloque_izquierda_col);
   
    //int id = threadIdx.x * dim + threadIdx.y;
    
    __shared__ int t_compartido[TESELAX * TESELAY];
    
    //Cargamos los valores de la tesela en memoria compartida
    for (int azulejo = 0; azulejo < (N + (TESELAX*TESELAY) - 1 / (TESELAX * TESELAY)); azulejo++)
    {
        if (fila < numFila && col < numCol && azulejo*(TESELAX) + threadIdx.x < numCol && azulejo*TESELAY + threadIdx.y < numFila)
        {
         //   printf("Hilo %d = %d\n", pos, t_compartido[id]);
            
            t_compartido[id] = dev_tablero[pos];
            
        }
    }
    __syncthreads();

    if (numFila > fila && numCol > col)
    {
        __syncthreads();
        int filaActual = pos / numCol;
        int colActual = pos - filaActual * numCol;
 
     //   printf("Valor t_compartido[%d],Valor dev_tablero=%d | Condicion hilo %d |Col anterior %d = [%d] > [%d]\n", t_compartido[id], dev_tablero[pos], pos, id, bloque_fila_anteriorY, (bloqueY));
        if (bloque_arriba_y == bloque_fila_actual ) //col < sig_bloqueX && fila < sig_bloqueY && col >= bloqueX && fila >= bloqueY && 
        {
            //printf("Hilo [%d] entra bloque_arriba = %d y bloquefila actual y col actual(%d, %d)\n", pos, bloque_arriba_y,  bloque_col_actual,bloque_fila_actual);
            //printf("Valor hilo %d [%d]--> id = %d e id_anterior = %d\n", pos, t_compartido[id],  id, (id - TESELAX));
            if (t_compartido[id] == -1) {
                //printf("Valor hilo %d = %d |t_compartido[id - TESELAX]\n", pos, t_compartido[id ],t_compartido[id - TESELAX]);
                if ((id - TESELAX) > 0)
                {
                 //   printf("Hollaaaa \n");
                    if (filaActual > 0 && filaActual <= numFila && t_compartido[id - TESELAX] != -1 && (id - TESELAX) > 0)
                    {
                        t_compartido[id] = t_compartido[id - TESELAX];
                        t_compartido[id - TESELAX] = -1;
                 //       printf("Hilo %d entra y carga valor del hilo %d = %d\n", pos, t_compartido[id]);
                        atomicAdd(&dev_index[0], 1);
                    }
                    else if (t_compartido[id - TESELAX] != -1)
                    {
                        hiprandState_t state;
                        hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
                        int color = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
               //         printf("COLOR %d\n", color);
                        t_compartido[id] = color;
                        atomicAdd(&dev_index[0], 1);
                    }
                    if (t_compartido[id - TESELAX] == -1)
                    {
                        dev_tablero[pos - numCol] = t_compartido[id - TESELAX];
                    }
                }
                else
                {
                    hiprandState_t state;
                    hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
                    int color = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
               //     printf("COLOR %d\n", color);
                    t_compartido[id] = color;
                    atomicAdd(&dev_index[0], 1);
                }
            }
            __syncthreads();
            
            dev_tablero[pos] = t_compartido[id];
        }
        else if (dev_tablero[pos] == -1)
        {
            if (filaActual > 0 && filaActual <= numFila && dev_tablero[pos - numCol] != -1)
            {
                dev_tablero[pos] = dev_tablero[pos - numCol];
                dev_tablero[pos - numCol] = -1;
                atomicAdd(&dev_index[0], 1);
            }
            else if (dev_tablero[pos - numCol] != -1)
            {
                hiprandState_t state;
                hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
                int color = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
        //        printf("COLOR %d\n", color);
                dev_tablero[pos] = color;
                atomicAdd(&dev_index[0], 1);
            }
        }
        
    }

}

__global__ void kernelEncontrarCaminos(int* dev_tablero, int numFila, int numCol, int* dev_index, int pos_encontrar, bool* dev_encontrado, int color)
{
    //Declaracion de varibles
    bool encontrado = false;
    bool camino_invalido = false;
    int posAux;
    int index = 0;

    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos

    int sig_bloqueX = (blockIdx.x + 1) * blockDim.x;            //Posicion inicial de la columna del siguiente hilo del siguiente bloque
    int sig_bloqueY = (blockIdx.y + 1) * blockDim.y;            //Posicion inicial de la fila del siguiente hilo en el siguiente bloque

    int bloqueX = blockIdx.x * blockDim.x;                      // //Posicion inicial de la columna del primer hilo del blqoue actual
    int bloqueY = blockIdx.y * blockDim.y;                      //Posicion inicial de la fila del primer hilo del blqoue actual

    int N = numFila;
    int dim = blockDim.x;
    int pos = ((col * N) + fila);
    int M = numCol;


    int bloque_fila_actual = (fila) / TESELAX;
    int bloque_col_actual = (col) / TESELAY;

    //Calculo del id del bloque tiene el hilo en la posicion de su izquierda, derecha, arriba y abajo
    int bloque_arriba_y = (fila - 1) / TESELAY;         //Arriba -1 Fila
    int bloque_abajo_y = (fila + 1) / TESELAY;         //Derecha +1
    int bloque_izquierda_x = (col - 1) / TESELAX;     //Abajo +1 Fila
    int bloque_derecha_x = (col + 1) / TESELAX;       //Izquierda -1

    //Recorrer 1� fila y 2�Columna del tablero en la que se encuentra la celda de POS
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;
    int ultima_posicion = pos;

   // printf("Hilo [%d] -divido %d / %d\n", pos, fila-1, TESELAY);

    if (numCol >= numFila)
    {
        N = numCol;
        M = numFila;
        dim = blockDim.y;
        pos = ((fila * N) + col);

        //Id bloque actual del hilo en X e Y
        bloque_fila_actual = (fila) / TESELAY;
        bloque_col_actual = (col) / TESELAX;

        bloque_arriba_y = (fila - 1) / TESELAY;         //Arriba -1 Fila
        bloque_abajo_y = (fila + 1) / TESELAY;         //Derecha +1
        bloque_izquierda_x = (col - 1) / TESELAX;     //Abajo +1 Fila
        bloque_derecha_x = (col + 1) / TESELAX;       //Izquierda -1

    }

    int id = threadIdx.y * dim + threadIdx.x;                   //Id del hilo dentro de la tesela
    int posId = id;
   // printf("Hilo [%d] - F[%d]C[%d]\n", pos, fila, col);
   // printf("Hilo [%d] - B (col, fila)=(%d, %d) Arriba[%d]\n", pos, bloque_col_actual, bloque_fila_actual, bloque_arriba_y);

//    printf("Hilo [%d] - Id Bloques: Fila Actual[%d] Arriba[%d], Col actual[%d]Izquierda[%d]\n", pos, bloque_fila_actual, bloque_arriba_y,bloque_col_actual, bloque_izquierda_col);

    //int id = threadIdx.x * dim + threadIdx.y;

    __shared__ int t_compartido[TESELAX * TESELAY];

    //Cargamos los valores de la tesela en memoria compartida
    for (int azulejo = 0; azulejo < (N + (TESELAX * TESELAY) - 1 / (TESELAX * TESELAY)); azulejo++)
    {
        if (fila < numFila && col < numCol && azulejo * (TESELAX)+threadIdx.x < numCol && azulejo * TESELAY + threadIdx.y < numFila)
        {
      //      printf("Hilo %d = %d\n", pos, t_compartido[id]);

            t_compartido[id] = dev_tablero[pos];

        }
    }
    __syncthreads();

    printf("\nAvanza a la pos IZQUIERDA [%d] hilo %d", posAux, pos);
    printf("\nIZQUIERDA COMPARTIDA [%d] - Pos shared %d", (posAux - numCol), (id - 1));
   // printf("\n T compartido en pos");

    if ((t_compartido[id] == color || t_compartido[id] == -1) && pos_encontrar == pos && numFila > fila && numCol > col)
    {
        printf("Hilo %d ha entrado a buscar camino [%d][%d]\n", pos, col, fila);
        encontrado = false;
        posAux = pos;
        posId= id;
        int tam_tesela = TESELAX * TESELAY;

        while ((posAux < (numCol * numFila)) && !camino_invalido && !encontrado)
        {
            bool uso_comp = false;
            int sigfila = (posAux + 1) / numCol;                 //Fila en la que se encuentra el siguiente elemento
            int sigcol = (posAux + 1) - sigfila * numCol;       //Columna en la que se encuentra el siguiente elemento

            int fila_anterior = (posAux - 1) / numCol;                 //Fila en la que se encuentra el elemento ANTERIOR
            int col_anterior = (posAux - 1) - fila_anterior * numCol; //Columna en la que se encuentra el elemento anterior

            int posSigFila = (posAux + numCol) / numCol;
            int fila_actual = posAux / numCol;
            int col_actual = posAux - fila_actual * numCol;

            //Comprobar si est� en memoria compartida o no
            printf("\nAvanza a la pos IZQUIERDA [%d] hilo %d", posAux, pos);
            printf("\nIZQUIERDA COMPARTIDA [%d] - Pos shared %d", (posAux - numCol), (id - 1));
            
            if (bloque_derecha_x == bloque_col_actual && (posId + 1) < tam_tesela && t_compartido[posId + 1] == color && sigcol > 0 && (posAux + 1) != ultima_posicion)    //col < sig_bloqueX && fila < sig_bloqueY && col >= bloqueX && fila >= bloqueY &&       //Nos desplazamos a la derecha )     //ABAJO memoria compartida
            {
                printf("\nCondicion DERECHA MEMORIA COMPARTIDA lleva a la posicion[%d] desde pos[%d] hilo %d\n", posAux + 1, pos, posId+1);
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux + 1;

                //Memoria compartida
                t_compartido[posId] = -1;
                posId = posId + 1;
                uso_comp = true;
                
            }
            else if (color == dev_tablero[posAux + 1] && sigcol > 0 && (posAux + 1) != ultima_posicion)          //Nos desplazamos a la derecha
            {
                printf("\nCondicion DERECHA lleva a la posicion[%d] desde pos[%d] hilo %d con color %d\n", posAux + 1, pos, posAux, color);
                index += 1;
                ultima_posicion = posAux;
                posAux += 1;
                dev_tablero[posAux] = -1;
            }
            else if (bloque_abajo_y == bloque_fila_actual && (posId + TESELAX) < tam_tesela && t_compartido[posId + TESELAX] == color && (posAux + numCol) < (numCol * numFila))  //ABAJO memoria compartida
            {
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux - numCol;
                printf("\nABAJO - MEM COMPARTIDA [%d]", (posAux + numCol));
                
                //Memoria compartida
                t_compartido[posId] = -1;
                posId = posId + TESELAX;
                uso_comp = true;
                
            }
            else if (color == dev_tablero[posAux + numCol] && (posAux + numCol) < (numCol * numFila) )  //Hacia abajo 
            {
                ultima_posicion = posAux;
                posAux = posAux + numCol;
                index += 1;
                dev_tablero[posAux] = -1;
                printf("\nAvanza a la pos de ABAJO [%d] ultima posicion %d hilo %d", posAux + numCol, posAux, pos); // && (id + 1) < tam_tesela) && (id - TESELAX) > 0))
                uso_comp = true;
            }
            else if (bloque_izquierda_x == bloque_col_actual && (posId - 1) > 0 && t_compartido[posId - 1] == color && col_anterior > 0 && (posAux - 1) != ultima_posicion)   //IZQUIERDA memoria compartida
            {
                printf("\nIZQUIERDA COMPARTIDA [%d] - Pos shared %d", (posAux - numCol), id - 1);
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux - numCol;
                uso_comp = true;
                //Memoria compartida
                t_compartido[posId] = -1;
                posId = posId - 1;
            }
            else if (color == dev_tablero[posAux - 1] && col_anterior > 0 && (posAux - 1) != ultima_posicion)           //Izquierda && (col_anterior < numCol - 1) 
            {
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux - 1;
                printf("\nAvanza a la pos IZQUIERDA [%d] hilo %d", posAux, pos);
                dev_tablero[posAux] = -1;
            }
            else if (bloque_arriba_y == bloque_fila_actual && (posId - TESELAX) > 0 && color == t_compartido[posId - TESELAX] && (posAux - numCol) >= 0 && filaActual >= 0 && filaActual <= numFila && (posAux - numCol) != ultima_posicion)    //Arriba memoria compartida
            {

                index += 1;
                ultima_posicion = posAux;
                printf("\nARRIBA Memoria compartida [%d] ultima posicion %d hilo %d", (posAux - numCol), id - TESELAX);
                posAux = posAux - numCol;

                //Memoria compartida
                t_compartido[posId] = - TESELAX;
                posId = posId - TESELAX;
                uso_comp = true;

            }
            else if (color == dev_tablero[posAux - numCol] && (posAux - numCol) >= 0 && filaActual >= 0 && filaActual <= numFila && (posAux - numCol) != ultima_posicion)  //ARRIBA
            {
                index += 1;
                ultima_posicion = posAux;
                printf("\nAvanza a la pos ARRIBA [%d] ultima posicion %d hilo %d", (posAux - numCol), ultima_posicion, pos);
                posAux = posAux - numCol;
                dev_tablero[posAux] = -1;
            }
            else
            {
                printf("\nNumero elementos %d\n", dev_index[0]);

                printf("\nCamino ENCONTRADO [%d]\n", pos);

                if (index > 0) {
                    atomicAdd(&dev_index[0], 1);
                    encontrado = true;
                }
                else {
                    encontrado = false;
                }

                printf("\nCamino no encontrado desde la posicion %d index vale %d\n", posAux, index);
                camino_invalido = true;
            }
            if (uso_comp == true) {
                dev_tablero[ultima_posicion] = t_compartido[id];
            }
        }
        
        dev_encontrado[0] = encontrado;
        printf("DEV_ENCONTRADO %d \n", dev_encontrado[0]);
        printf("DEV_INDEX %d \n", dev_index[0]);
        if (dev_index[0] >= 1 && pos == pos_encontrar)
        {
            printf("Posicion a encontrar %d \n", pos_encontrar);
            dev_tablero[pos_encontrar] = -1;              //Establecemos la posicion a encontrar en -1
        }
    }
    __syncthreads();

}

__global__ void kernelEncontrarBomba(int* dev_tablero, int numFila, int numCol, int pos_encontrar, int* dev_index_fila, int* dev_index_col)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int sig_bloqueX = (blockIdx.x + 1) * blockDim.x;
    int sig_bloqueY = (blockIdx.y + 1) * blockDim.y;
    int bloqueX = blockIdx.x * blockDim.x;
    int bloqueY = blockIdx.y * blockDim.y;
    int N = numFila;
    int dim = blockDim.x;

    int pos = ((col * N) + fila);
    if (numCol > numFila)
    {
        N = numCol;
        dim = blockDim.y;
        pos = ((fila * N) + col);
        //id = blockIdx.y * dim + threadIdx.y;
    }
    int id = threadIdx.x * dim + threadIdx.y;
    id = threadIdx.y * dim + threadIdx.x;
    int tamX = numFila;

  //  printf(" F[%d] C[%d] dim(%d, %d) | id = %d | pos = %d\n", fila, col, numFila, numCol, id, pos);

    __shared__ int t_compartido[2 * 5];

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna de la posici�n a encontrar
    int filaEncontrar = pos_encontrar / numCol;
    int colEncontrar = pos_encontrar - filaEncontrar * numCol;

    if (filaActual == filaEncontrar && (int)dev_index_fila > 5 && dev_tablero[pos] == -1 && numFila > fila && numCol > col)
    {
        atomicAdd(&dev_index_fila[0], 1);
    }

    if (colActual == colEncontrar && (int)dev_index_col > 5 && dev_tablero[pos] == -1 && numFila > fila && numCol > col)
    {
        atomicAdd(&dev_index_col[0], 1);
    }

    __syncthreads();
    if (dev_index_fila[0] != dev_index_col[0] && (numFila > col) && (numCol > fila))
    {
        //  printf("Valor del contador de fila %d y contador columna %d \n", dev_index_col[0], dev_index_fila[0]);
        if ((dev_index_fila[0] == 5 && dev_index_col[0] == 1) || (dev_index_col[0] == 5 && dev_index_fila[0] == 1))
        {
            dev_tablero[pos_encontrar] = 'B';
        }
    }
}

__global__ void kernelEncontrarRompecabezasTNT(int* dev_tablero, int numFila, int numCol, int pos_encontrar, int* dev_index_RC, int dev_semilla, int dificultad)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Posicion en la que nos encontramos
    int sig_bloqueX = (blockIdx.x + 1) * blockDim.x;
    int sig_bloqueY = (blockIdx.y + 1) * blockDim.y;
    int bloqueX = blockIdx.x * blockDim.x;
    int bloqueY = blockIdx.y * blockDim.y;
    int N = numFila;
    int dim = blockDim.x;

    int pos = ((col * N) + fila);
    if (numCol > numFila)
    {
        N = numCol;
        dim = blockDim.y;
        pos = ((fila * N) + col);
    }
    int id = threadIdx.x * dim + threadIdx.y;
    id = threadIdx.y * dim + threadIdx.x;
    int tamX = numFila;

   // printf(" F[%d] C[%d] dim(%d, %d) | id = %d | pos = %d\n", fila, col, numFila, numCol, id, pos);

    __shared__ int t_compartido[2 * 5];

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna de la posici�n a encontrar
    int filaEncontrar = pos_encontrar / numCol;
    int colEncontrar = pos_encontrar - filaEncontrar * numCol;

    printf("hilo %d valor %d\n", pos, dev_tablero[pos]);
    if (dev_tablero[pos] == -1 && numFila > fila && numCol > col)
    {
        printf("Entro hilo %d e incremento %d\n", pos, dev_index_RC[0]);
        atomicAdd(&dev_index_RC[0], 1);
    }
    __syncthreads();

    printf("Contador TNT - RC %d\n", dev_index_RC[0]);
    if (dev_index_RC[0] == 6 && pos == pos_encontrar)
    {
        dev_tablero[pos_encontrar] = 'T';
        dev_index_RC[0] = 0;
    }
    else if (dev_index_RC[0] >= 7 && pos == pos_encontrar)
    {
        hiprandState_t state;
        hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
        int color = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
        int colorS = 7 + color;
        dev_tablero[pos_encontrar] = colorS;
        dev_index_RC[0] = 0;
    }

}



//Inicializamos el tablero
int* inicializarTablero(int* h_tablero, int size, int numCol, int numFila, int dificultad, int hilosBloqueX, int hilosBloqueY, int gridX, int gridY)
{
    int* (dev_Tablero);

    //Reservar espacio en memoria para GPU (2 matrices y matriz resultado)
    hipMalloc((void**)&dev_Tablero, size * sizeof(int));

    //Copiamos datos a la GPU 
    hipMemcpy(dev_Tablero, h_tablero, size * sizeof(int), hipMemcpyHostToDevice);

    unsigned int semilla = time(NULL);
    dim3 dimGrid(gridX, gridY);
    dim3 dimBlock(hilosBloqueX, hilosBloqueY);
    kernelGenerarTablero << <dimGrid, dimBlock >> > (dev_Tablero, semilla, dificultad, numCol, numFila, hilosBloqueX, hilosBloqueY);

    // Copiamos de la GPU a la CPU
    hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);

    return h_tablero;

}

//Funcion que llama a kernel para encontrar todos los caminos hacia bloque indicado
int encontrarCamino(int* h_tablero_original, int numFilas, int numColumnas, int coordX, int coordY, int dificultad, int vida, int hilosBloqueX, int hilosBloqueY, int gridX, int gridY)
{
    int* h_tablero = h_tablero_original;
    int* (dev_Tablero), * (dev_index), * (dev_index_fila), * (dev_index_col), * (dev_index_RC), *(dev_hilos_x);
    bool* dev_encontrado;
    int size = numFilas * numColumnas;
    bool h_encontrado = true;
    int* h_index = { 0 };
    int* h_index_col = { 0 };
    int* h_index_fila = { 0 };
    int* h_index_RC = { 0 };

    //int pos_encontrar = coordX * numFilas + coordY;   //Posicion a ENCONTRAR en el vector 1D
    int  pos_encontrar = coordX * numColumnas + coordY;
    if (numColumnas > numFilas)
    {
        pos_encontrar = coordX * numColumnas + coordY;   //Posicion a ENCONTRAR en el vector 1D   
    }

    int color = h_tablero[pos_encontrar];

    unsigned int semilla = time(NULL);
    printf("Posicion a ENCONTRAR %d\n", pos_encontrar);
    //Reservar espacio en memoria para GPU (2 matrices y matriz resultado)
    hipMalloc((void**)&dev_Tablero, size * sizeof(int));
    hipMalloc((void**)&dev_index, sizeof(int));
    hipMalloc((void**)&dev_index_col, sizeof(int));
    hipMalloc((void**)&dev_index_fila, sizeof(int));
    hipMalloc((void**)&dev_index_RC, sizeof(int));
    hipMalloc((void**)&dev_hilos_x, sizeof(int));
    hipMalloc(&dev_encontrado, sizeof(bool));

    //Copiamos datos a la GPU 
    hipMemcpy(dev_Tablero, h_tablero, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index, h_index, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index, h_index, sizeof(int), hipMemcpyHostToDevice);

    hipMemcpy(dev_index_col, h_index_col, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_fila, h_index_fila, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_RC, h_index_RC, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(hiloX), &hilosBloqueX, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_encontrado, &h_encontrado, sizeof(bool), hipMemcpyHostToDevice);


    dim3 dimGrid(gridX, gridY);
    dim3 dimBlock(hilosBloqueX, hilosBloqueY);

    //Segun si es alguno de los bloques especiales o es una jugada normal (66 --> B, 84 --> T,)
    int contenido = h_tablero[pos_encontrar];

    if (contenido == 'B')
    {
        kernelBomba << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }

    else if (contenido == 'T')
    {
        kernelTNT << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }

    else if (7 <= contenido && contenido <= 13) //Si es RC
    {
        int colorBorrar = contenido % 7;
        kernelRompeCabezas << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, colorBorrar, pos_encontrar);
        printf("COLOrrr %d \n", colorBorrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }
    else //Si es bloque simple
    {
        int cont = 0;
        //Desde posici�n idicada se encuentran todos los caminos con el mismo color
        while (cont < numColumnas * numFilas)
        {
            printf("contador %d \n", cont);
            while (h_encontrado)
            {

                kernelEncontrarCaminos << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, dev_index, pos_encontrar, dev_encontrado, color);
                hipMemcpy(&h_encontrado, dev_encontrado, sizeof(bool), hipMemcpyDeviceToHost);
                hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&h_index, dev_index, sizeof(int), hipMemcpyDeviceToHost);
                printf("Valor del puntero %d \n", h_encontrado);
                printf("H_inxex %d\n", h_index);
                //mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
            }

            if (h_tablero[cont] == -1)
            {
                kernelEncontrarCaminos << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, dev_index, cont, dev_encontrado, color);
                hipMemcpy(&h_encontrado, dev_encontrado, sizeof(bool), hipMemcpyDeviceToHost);
                hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&h_index, dev_index, sizeof(int), hipMemcpyDeviceToHost);
                // mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
            }
            cont += 1;
        }
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
        if ((int)h_index == 0 && vida >= 1)
        {
            vida = vida - 1;
        }
        h_index_fila = { 0 };
        h_index_col = { 0 };
        hipMemcpy(dev_index_col, h_index_col, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(dev_index_fila, h_index_fila, sizeof(int), hipMemcpyHostToDevice);
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
        kernelEncontrarBomba << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar, dev_index_fila, dev_index_col);
        hipMemcpy(&h_index_fila, dev_index_fila, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_index_col, dev_index_col, sizeof(int), hipMemcpyDeviceToHost);

        printf("N Filas %d - N Columnas %d \n", h_index_fila, h_index_col);
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
        h_index_RC = { 0 };
        hipMemcpy(dev_index_RC, h_index_RC, sizeof(int), hipMemcpyHostToDevice);
        kernelEncontrarRompecabezasTNT << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar, dev_index_RC, semilla, dificultad);
        hipMemcpy(&h_index_RC, dev_index_RC, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);

    }
    mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);


    h_index = { 0 };
    int iteraciones = 10;
    //Bucle para reemplazar las posiciones eliminadas mientras que se pueda hacer algun cambio y si no termine
    while (iteraciones > 0)
    {
        kernelReemplazarPosiciones << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, semilla, dificultad, dev_index);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_index, dev_index, sizeof(int), hipMemcpyDeviceToHost);
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
        iteraciones = (int)h_index;
    }
    mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);

    hipFree(dev_encontrado);
    hipFree(dev_Tablero);
    hipFree(dev_index);
    hipFree(dev_index_fila);
    hipFree(dev_index_col);
    hipFree(dev_index_RC);
    return vida;
}

void main(int argc, char* argv[])
{
    //Declaracion variables
    //int* h_tablero;
    int numFilas = 3;
    int numColumnas = 9;
    int coordenadaX;
    int coordenadaY;
    int size = numFilas * numColumnas;
    int dificultad = 4;
    bool terminado = false;
    int vida = 5;
    char modoJuego = 'A';
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    char* nombre = deviceProp.name;
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    int maxThreadsSM = deviceProp.maxThreadsPerMultiProcessor;
    int maxBlockx = deviceProp.maxGridSize[0];
    int maxBlocky = deviceProp.maxGridSize[1];
    int maxGridX = deviceProp.maxThreadsDim[0];
    int maxGridY = deviceProp.maxThreadsDim[1];

    printf("Nombre del Device: %s\n", deviceProp.name);
    printf("Num maximo de hilos por bloque: %d\n", deviceProp.maxThreadsPerBlock);
    printf("Num maximo de bloque s: %d\n", maxBlockx);
    printf("Dimensiones maximas para organizar los hilos en bloques (%d, %d, %d):\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("Dimensiones maximas para organizar los bloques en el grid (%d, %d, %d):\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);

    /*
    * int hilosBloqueX = ceil(numFilas / (float)2);
    int hilosBloqueY = ceil(numColumnas / (float)2);
    int gridX = ceil(numFilas / (float)hilosBloqueX);
    int gridY = ceil(numColumnas / (float)hilosBloqueY);
    if (numColumnas > numFilas) {
        hilosBloqueX = ceil(numColumnas / (float)2);
        hilosBloqueY = ceil(numFilas / (float)2);
        gridX = ceil(numColumnas / (float)hilosBloqueX);
        gridY = ceil(numFilas / (float)hilosBloqueY);
    }
    */
    
    int hilosBloqueX = ceil(numColumnas / (float)2);
    int hilosBloqueY = ceil(numFilas / (float)2);
    int gridX = ceil(numColumnas / (float)hilosBloqueX);
    int gridY = ceil(numFilas / (float)hilosBloqueY);

    /*
    if (numColumnas > numFilas) {
        hilosBloqueX = ceil(numFilas / (float)2);
        hilosBloqueY = ceil(numColumnas / (float)2);
        gridX = ceil(numFilas / (float)hilosBloqueX);
        gridY = ceil(numColumnas / (float)hilosBloqueY);
    }
    */
    printf("dimBlock(%d, %d), dimGrid(%d, %d): ", hilosBloqueX, hilosBloqueY, gridX, gridY);
    //Pasamos a memoria constante el numero de filas y columnas introducidas por el usuario
    hipMemcpyToSymbol(HIP_SYMBOL(FILAS), &numFilas, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(COLUMNAS), &numColumnas, sizeof(int));

    //Reservamos memoria para el tablero, ya que no esta inicializado
  //  int* h_tablero = (int*)malloc(numFilas * numColumnas * sizeof(int));

    //Llamamos a la funcion que inicializa con valores aleatorios el tablero
 //   h_tablero = inicializarTablero(h_tablero, size, numColumnas, numFilas, dificultad, hilosBloqueX, hilosBloqueY, gridX, gridY);
   // int h_tablero[16] = { 3,3,3,3,4,3,3,4,3,1,4,3,'B',3,1,3};
   //int h_tablero[27] = { 3,3,3,3,3,3,3,4,4,4,4,3,1,3,3,3,4,3,3,3,4,3,3,4,3,4,4 };
    int h_tablero[27] = { 3,3,3,3,3,3,3,4,4,4,4,3,1,'B',3,3,4,3,3,3,4,3,3,4,3,4,4};
    // int h_tablero[25] = { 3,2,1,5,5,3,3,6,7,3,9,3,'B',3,1,3,1,3,3,3,4,1,1,4,3 };
     //Mostramos el tablero



     //Codigo para ejecutar programa y recibir datos por comando
     //Controla que no de error la llamada
    if (argc == 1)  //No se ha ejecutado por comando
    {
        printf("\nElija el modo de juego: A (Automatico) - M (Manual):  \n");
        scanf("%c", &modoJuego);
        printf("Modo de juego seleccionado: %c \n", modoJuego);
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
    }

    else if (argc == -1)
    {
        printf("ERROR en ejecucion\n");
    }
    //Controla que tengamos los argumentso encesarios (tipo de ejecucion, dificultad, filas, columnas)
    else if (argc < 5)
    {
        printf("ERROR: faltan argumentos de entrada\n");
        printf("%d \n", argc);
    }
    //Controlan que no se pasen mas argumentos de los deseados
    else if (argc > 5)
    {
        printf("ERROR: sobran argumentos de entrada\n");
    }
    else //Realiza llamada con los 4 argumemtos
    {
        //Guarda los argumentos pasadas en las respectivas variables
        modoJuego = (char)argv[1];
        dificultad = std::stoi(argv[2]);    //Guarda valor argumentos usando funcion stoi para convertirlo a int
        numFilas = std::stoi(argv[3]);
        numColumnas = std::stoi(argv[4]);

    }
    if (hilosBloqueX >= maxBlockx && hilosBloqueY >= maxBlocky && gridX > maxGridX && gridY > maxGridY)
    {
        printf("\nSe sobrepasan las dimensiones asociadas a la tarjeta grafica  \n");
    }
    else
    {
        while (vida > 0)
        {
            
            
           
            if (modoJuego == 'M' || modoJuego == 'm')
            {
                printf("\nIntroduzca las coordenadas del bloque que desea eliminar (x, y):  \n");
                scanf("%d %d", &coordenadaX, &coordenadaY);
            }
            else if (modoJuego == 'A' || modoJuego == 'a')
            {
                coordenadaX = (rand() % numFilas);
                coordenadaY = (rand() % numColumnas);
                printf("\nCoordenadas (%d, %d)  \n", coordenadaX, coordenadaY);
            }

            if ((coordenadaX < numFilas) && (coordenadaY < numColumnas) && (coordenadaX >= 0) && (coordenadaY >= 0))
            {
                vida = encontrarCamino(h_tablero, numFilas, numColumnas, coordenadaX, coordenadaY, dificultad, vida, hilosBloqueX, hilosBloqueY, gridX, gridY);
                printf("\nVida restante: %d \n", vida);
            }
            else
            {
                printf("\nLas coordenadas introducidas se encuentran fuera del rango del tablero [%d][%d] \n", numFilas, numColumnas);
            }
           


        }
         
        printf("\nPERDEDOR \n");
    }

}



