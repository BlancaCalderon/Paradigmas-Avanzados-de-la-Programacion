#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>



//Genera una semilla aleatoria para cada hilo
int generarSemilla()
{
    hiprandState_t state;
    int semilla = time(NULL) * 3663828372 + 12345 + rand();
    return semilla;
}

//Funcion que muestra el tablero por consola
void mostrarTablero(int* tablero, int numFilas, int numColumnas, int dificultad)
{
    int N = numFilas;
    int M = numColumnas;

    if (numFilas > numColumnas || numColumnas > numFilas) {     //Calculo realizado para mostrar correctamente los tableros ASIMETRICOS, solo entra en el caso de que el numero de columnas sea mayor
        N = numColumnas;
        M = numFilas;
    }
    printf("Tablero \n");
    for (int i = 0; i < numFilas; i++)                          //Recorremos las filas del tablero
    {
        printf(" \n");
        for (int j = 0; j < numColumnas; j++)                   //Recorremos las columnas del tableros
        {
            int num = tablero[i * N + j];                       //Calculamos su posicion correspondiente en el vector 1D, N ser� el numero de filas si numFilas > numColumnas, y N ser� el n�mero de columnas en el caso contrario.
            if (num > dificultad)
            {
                if (7 <= num && num <= 13)                      //Como nuestro tablero es de tipo int los rompecabezas se encuentran entre un rango de 7 y 13 (7 asociado al rompecabezas y el resto al numero aleatorio generado en el kernel TNT)
                {
                    printf("  RC%d ||", num % 7);           //Modulo 7 nos devuelve el numero aleatorio generado en el kernel
                }
                else
                {
                    printf("  %c   ||", (char)num);             //Si otro numero se corresponde con su valor en ASCII
                }
            }
            else
            {
                printf("  %d   ||", num);                       //Numero del tablero
            }
        }
        printf("\n");
        printf(" \n");
    }
    printf("\n");
}

//Kernel que elimina la fila y columna de la posicion pasada (Bomba)
__global__ void kernelBomba(int* dev_tablero, int numFila, int numCol, int pos_encontrar)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;         //Columna del hilo en el tablero
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Fila del hilo en el tablero
    int N = numFila;
    int pos = ((col * N) + fila);                              //Posicion del hilo en el vector 1D

    if (numCol > numFila)                                      //Controlamos el calculo de la posicion por si nos llega una matriz asimetrica en la que el numero de columnas es mayor que e de filas
    {
        N = numCol;
        pos = ((fila * numCol) + col);                         //El calculo de la posicion variara en funcion de las dimensiones de la matriz
    }

    //Calcula fila y columna a borrar
    int filaBorrar = pos_encontrar / numCol;                //Calculamos la fila en la que se encuentra la posicion a encontrar
    int colBorrar = pos_encontrar - filaBorrar * numCol;    //Calculamos la columna en la que se encuentra la posicion a encontrar

    //Comprobamos i el hilo esta en la fila o columna que queremos borrar
    if (filaBorrar == fila || colBorrar == col && 0 < fila && fila < numFila && 0 < col && col < numCol)
    {
        dev_tablero[pos] = -1;                              //Indicamos que se borra
    }
    dev_tablero[pos_encontrar] = -1;                        //Eliminamos bloque especial
}


//Kernel que elimina los elementos adyacentes a una posici�n (radio 4 elementos) (TNT)
__global__ void kernelTNT(int* dev_tablero, int numFila, int numCol, int pos_encontrar)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;         //Columna del hilo en el tablero
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Fila del hilo en el tablero
    int N = numFila;
    int pos = ((col * N) + fila);                              //Posicion del hilo en el vector 1D

    if (numCol > numFila)                                      //Controlamos el calculo de la posicion por si nos llega una matriz asimetrica en la que el numero de columnas es mayor que e de filas
    {
        N = numCol;
        pos = ((fila * numCol) + col);                         //El calculo de la posicion variara en funcion de las dimensiones de la matriz
    }

    if (numFila > fila && numCol > col)                        //Comprobamos que el hilo que nos llega este dentro de las dimensiones de la matriz
    {

        //Calcula fila y columna a borrar teniendo en cuenta el rango
        int filaBorrar = pos_encontrar / numCol;
        int colBorrar = pos_encontrar - filaBorrar * numCol;

        int filaBorrarDer = filaBorrar + 4;
        int colBorrarAbajo = colBorrar + 4;
        int filaBorrarIzq = filaBorrar - 4;
        int colBorrarArriba = colBorrar - 4;

        //si posici�n actual es adyacente y esta dentro del rango que queremos borrar (4)
        if (filaBorrarIzq <= fila && fila <= filaBorrarDer && colBorrarArriba <= col && col <= colBorrarAbajo && 0 <= fila && fila < numFila && 0 <= col && col < numCol && pos < (numCol * numFila))
        {
            dev_tablero[pos] = -1; //Indicamos que se borra
        }
    }
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial

}


//Kernel que elimina todos las posiciones del color indicado (ROMPECABEZAS)
__global__ void kernelRompeCabezas(int* dev_tablero, int numFila, int numCol, int color, int pos_encontrar)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;         //Columna del hilo en el tablero
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Fila del hilo en el tablero
    int N = numFila;
    int pos = ((col * N) + fila);                              //Posicion del hilo en el vector 1D

    if (numCol > numFila)                                      //Controlamos el calculo de la posicion por si nos llega una matriz asimetrica en la que el numero de columnas es mayor que e de filas
    {
        N = numCol;
        pos = ((fila * numCol) + col);                         //El calculo de la posicion variara en funcion de las dimensiones de la matriz
    }

    if (numFila > fila && numCol > col)                        //Comprobamos que el hilo que nos llega este dentro de las dimensiones de la matriz
    {

        //si posici�n actual tiene el color indicado se elimina
        if (dev_tablero[pos] == color && pos < (numCol * numFila))  //Si la posicion es igual al color de la posicion a encontrar 
        {
            dev_tablero[pos] = -1;      //Indicamos que se borra
        }
    }
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial

}


//Kernel que lleva a cabo la generacion del tablero de forma aleatoria
__global__ void kernelGenerarTablero(int* dev_tablero, int dev_semilla, int dificultad, int numCol, int numFila)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;         //Columna del hilo en el tablero
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Fila del hilo en el tablero
    int N = numFila;
    int pos = ((col * N) + fila);                              //Posicion del hilo en el vector 1D

    if (numCol > numFila)                                      //Controlamos el calculo de la posicion por si nos llega una matriz asimetrica en la que el numero de columnas es mayor que e de filas
    {
        N = numCol;
        pos = ((fila * numCol) + col);                         //El calculo de la posicion variara en funcion de las dimensiones de la matriz
    }

    if (numFila > fila && numCol > col)                        //Comprobamos que el hilo que nos llega este dentro de las dimensiones de la matriz
    {
        hiprandState_t state;
        hiprand_init(dev_semilla, pos, 0, &state);                         //Genera diferentes secuencias de numeros aleatorio a partir de la semlla
        dev_tablero[pos] = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
    }
}

__global__ void kernelReemplazarPosiciones(int* dev_tablero, int numFila, int numCol, int dev_semilla, int dificultad, int* dev_index)
{
    dev_index[0] = 0;                                          //Lo utilizamos para contabilizar el numero de llamadas que hay que realizar al kernel 
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;         //Columna del hilo en el tablero
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Fila del hilo en el tablero
    int N = numFila;
    int pos = ((col * N) + fila);                              //Posicion del hilo en el vector 1D

    if (numCol > numFila)                                      //Controlamos el calculo de la posicion por si nos llega una matriz asimetrica en la que el numero de columnas es mayor que e de filas
    {
        N = numCol;
        pos = ((fila * numCol) + col);                         //El calculo de la posicion variara en funcion de las dimensiones de la matriz
    }

    if (dev_tablero[pos] == -1 && numFila > fila && numCol > col)   //Comprobamos que la posicion que nos llega sea -1 en el tablero y que el hiloeste dentro de las dimensiones de la matriz
    {

        int filaActual = pos / numCol;
        int colActual = pos - filaActual * numCol;
        if (filaActual > 0 && filaActual <= numFila && dev_tablero[pos - numCol] != -1)  //Si la posicion de arriba es distinta de -1 se la asignamos a la posicion que nos llega y se la quitamos a la de arriba
        {
            dev_tablero[pos] = dev_tablero[pos - numCol];               //Le asignamos el valor a la posicion del hilo
            dev_tablero[pos - numCol] = -1;                             //Establecemos el valor de la posicion de arriba en -1
            atomicAdd(&dev_index[0], 1);                                //Incrementamos el valor del contador, el cual sera utilizado para parar el bucle while que llama a este kernel, ya que si vale 1 significa que no quedan posiciones con valor -1
        }
        else if (dev_tablero[pos - numCol] != -1)
        {
            hiprandState_t state;
            hiprand_init(dev_semilla, pos, 0, &state);                   //Genera diferentes secuencias de numeros aleatorio a partir de la semilla
            int color = abs((int)(hiprand(&state) % dificultad) + 1);    //Rellena tablero con numeros aleatorios entre 1 y dificultad
            dev_tablero[pos] = color;                                   //Asignamos un nuevo color a la posicion del tablero
            atomicAdd(&dev_index[0], 1);
        }
    }

}

__global__ void kernelEncontrarCaminos(int* dev_tablero, int numFila, int numCol, int* dev_index, int pos_encontrar, bool* dev_encontrado, int color)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;         //Columna del hilo en el tablero
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Fila del hilo en el tablero
    int N = numFila;
    int pos = ((col * N) + fila);                              //Posicion del hilo en el vector 1D

    if (numCol > numFila)                                      //Controlamos el calculo de la posicion por si nos llega una matriz asimetrica en la que el numero de columnas es mayor que e de filas
    {
        N = numCol;
        pos = ((fila * numCol) + col);                         //El calculo de la posicion variara en funcion de las dimensiones de la matriz
    }

    bool encontrado = false;
    bool camino_invalido = false;
    int posAux;
    int index = 0;

    //Recorrer 1� fila y 2�Columna del tablero en la que se encuentra la celda de POS
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;
    int ultima_posicion = pos;

    if ((dev_tablero[pos] == color || dev_tablero[pos] == -1) && pos_encontrar == pos)      //Comprobamos que la posicion que nos llega se corresponda con la posicion a encontrar y que su valor sea el color o -1 (lo que significa que ya ha formado parte de un camino pero puede haber mas)
    {
        encontrado = false;
        posAux = pos;

        while ((posAux < numCol * numFila) && !encontrado && !camino_invalido)              //Finaliza cuando no encuentra un camino posible
        {
            //Variables que utilizamos para ver que no nos salimos del rango de la matriz segun vamos incrementando el valor de posAux
            int sigfila = (posAux + 1) / numCol;                                    //Fila en la que se encuentra el siguiente elemento
            int sigcol = (posAux + 1) - sigfila * numCol;                           //Columna en la que se encuentra el siguiente elemento

            int fila_anterior = (posAux - 1) / numCol;                              //Fila en la que se encuentra el elemento ANTERIOR
            int col_anterior = (posAux - 1) - fila_anterior * numCol;               //Columna en la que se encuentra el elemento anterior

            int posSigFila = (posAux + numCol) / numCol;
            int fila_actual = posAux / numCol;
            int col_actual = posAux - fila_actual * numCol;

            if (color == dev_tablero[posAux + 1] && sigcol > 0 && (posAux + 1) != ultima_posicion)                                           //Comprueba el valor de su posicion DERECHA
            {
                index += 1;                 //Incrementa el indice pasa saber que hay un camino
                ultima_posicion = posAux;   //Almacenamos la ultima posicion
                posAux += 1;                //Avanzamos a la derecha
                dev_tablero[posAux] = -1;   //Marcamos la posicion del tablero a -1
            }
            else if (color == dev_tablero[posAux + numCol] && (posAux + numCol) != ultima_posicion && (posAux + numCol) < numCol * numFila)   //Comprueba el color de ABAJO
            {
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux + numCol;       //Avanzamos abajo
                dev_tablero[posAux] = -1;
            }
            else if (color == dev_tablero[posAux - 1] && col_anterior > 0 && (posAux - 1) != ultima_posicion)                                  //Comprueba el color de posicion a la IZQUIERDA
            {
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux - 1;            //Avanzamos a la izquierda
                dev_tablero[posAux] = -1;
            }
            else if (color == dev_tablero[posAux - numCol] && (posAux - numCol) != ultima_posicion && (posAux - numCol) >= 0 && filaActual > 0 && filaActual < numFila)  //Comprueba el color de su posicion ARRIBA
            {
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux - numCol;       //Avanzamos arriba
                dev_tablero[posAux] = -1;
            }
            else
            {
                if (index > 0) {                     //Si el indice es mayor de 0 significa que en una iteracion anterior del bucle while ha encontrado un camino
                    atomicAdd(&dev_index[0], 1);     //Incrementamos una variable que sera devuelta al host para saber si se ha encontrado un camino
                    encontrado = true;               //Volvera al Host, y si es true seguira en el bucle while para ver si hay mas caminos que no ha recorrido
                }
                else
                {
                    encontrado = false;
                }

                camino_invalido = true;              //Para el bucle del device, es necesario ya que que encontrado = true cuando el hilo a encontrado algun camino
            }

        }

    }
    dev_encontrado[0] = encontrado;                 //Almacena el valor de booleano encontrado y lo devuelve al host

    if (dev_index[0] >= 1)
    {
        dev_tablero[pos_encontrar] = -1;              //Establecemos la posicion a encontrar en -1
    }

}

/* Kernel para encontrar bomba */
__global__ void kernelEncontrarBomba(int* dev_tablero, int numFila, int numCol, int pos_encontrar, int* dev_index_fila, int* dev_index_col)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;         //Columna del hilo en el tablero
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Fila del hilo en el tablero
    int N = numFila;
    int pos = ((col * N) + fila);                              //Posicion del hilo en el vector 1D

    if (numCol > numFila)                                      //Controlamos el calculo de la posicion por si nos llega una matriz asimetrica en la que el numero de columnas es mayor que e de filas
    {
        N = numCol;
        pos = ((fila * numCol) + col);                         //El calculo de la posicion variara en funcion de las dimensiones de la matriz
    }

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna de la posici�n a encontrar
    int filaEncontrar = pos_encontrar / numCol;
    int colEncontrar = pos_encontrar - filaEncontrar * numCol;

    if (filaActual == filaEncontrar && (int)dev_index_fila < 5 && dev_tablero[pos] == -1 && (numFila > fila) && (numCol > col))//Si el hilo que llega se encuentra en la misma FILA que la de las coordendas introducidas por el usuario 
    {
        atomicAdd(&dev_index_fila[0], 1);       //Incrementamos el contador de las filas
    }

    if (colActual == colEncontrar && (int)dev_index_col < 5 && dev_tablero[pos] == -1 && (numFila > col) && (numCol > fila)) //Si el hilo que llega se encuentra en la misma COLUMNA que la de las coordendas introducidas por el usuario 
    {
        atomicAdd(&dev_index_col[0], 1);        //Incrementamos el contador de las columnas
    }


    if (dev_index_fila[0] != dev_index_col[0] && (numFila > col) && (numCol > fila))     //Comprobamos que sean distintos, ya que si son iguales no cumpliria con nuestro requisito, solo puede haber 5 seguidos en una fila o en una columna, pero no simultaneamente
    {
        if ((dev_index_fila[0] == 5 && dev_index_col[0] == 1) || (dev_index_col[0] == 5 && dev_index_fila[0] == 1))  //Comprobamos que uno de los indices a encontrar sea 5
        {
            dev_tablero[pos_encontrar] = 'B';    //Generamos la bomba en el tablero
            dev_index_fila[0] = 0;
            dev_index_col[0] = 0;
        }
    }

}

/* Kernel que genera un Rompecabezas RC y un TNT */
__global__ void kernelEncontrarRompecabezasTNT(int* dev_tablero, int numFila, int numCol, int pos_encontrar, int* dev_index_RC, int dev_semilla, int dificultad)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x;         //Columna del hilo en el tablero
    int fila = (blockIdx.y * blockDim.y) + threadIdx.y;        //Fila del hilo en el tablero
    int N = numFila;
    int pos = ((col * N) + fila);                              //Posicion del hilo en el vector 1D

    if (numCol > numFila)                                      //Controlamos el calculo de la posicion por si nos llega una matriz asimetrica en la que el numero de columnas es mayor que e de filas
    {
        N = numCol;
        pos = ((fila * numCol) + col);                         //El calculo de la posicion variara en funcion de las dimensiones de la matriz
    }

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna de la posici�n a encontrar
    int filaEncontrar = pos_encontrar / numCol;
    int colEncontrar = pos_encontrar - filaEncontrar * numCol;

    if (dev_tablero[pos] == -1)                                 //Si la posicion que nos llega es un -1
    {
        atomicAdd(&dev_index_RC[0], 1);                         //Incrementamos el contador mediante una variable atomica
    }
    __syncthreads();

    if (dev_index_RC[0] == 6 && pos == pos_encontrar)           //Si el indice vale 6 es el TNT
    {
        dev_tablero[pos_encontrar] = 'T';
        dev_index_RC[0] = 0;
    }
    else if (dev_index_RC[0] >= 7 && pos == pos_encontrar)      //Si el indice es mayor de o igual de 7 introducimos un RC
    {
        hiprandState_t state;
        hiprand_init(dev_semilla, pos, 0, &state);                   //Genera diferentes secuencias de numeros aleatorio a partir de la semilla
        int color = abs((int)(hiprand(&state) % dificultad) + 1);    //Rellena tablero con numeros aleatorios entre 1 y la dificultad                                 
        dev_tablero[pos_encontrar] = 7 + color;                      //Introducimos el RC en el tablero mas el color
        dev_index_RC[0] = 0;
    }

}

//Inicializamos el tablero
int* inicializarTablero(int* h_tablero, int size, int numCol, int numFila, int dificultad, int hilosBloqueX, int hilosBloqueY, int gridX, int gridY)
{
    int* (dev_Tablero);

    //Reservar espacio en memoria para GPU (2 matrices y matriz resultado)
    hipMalloc((void**)&dev_Tablero, size * sizeof(int));

    //Copiamos datos a la GPU 
    hipMemcpy(dev_Tablero, h_tablero, size * sizeof(int), hipMemcpyHostToDevice);

    int semilla = generarSemilla();
    dim3 dimGrid(gridX, gridY);
    dim3 dimBlock(hilosBloqueX, hilosBloqueY);
    kernelGenerarTablero << <dimGrid, dimBlock >> > (dev_Tablero, semilla, dificultad, numCol, numFila);

    // Copiamos de la GPU a la CPU
    hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);

    return h_tablero;

}

//Funcion que llama a kernel para encontrar todos los caminos hacia bloque indicado
int jugar(int* h_tablero_original, int numFilas, int numColumnas, int coordX, int coordY, int dificultad, int vida, int hilosBloqueX, int hilosBloqueY, int gridX, int gridY)
{
    int* h_tablero = h_tablero_original;
    int* (dev_Tablero), * (dev_index), * (dev_index_fila), * (dev_index_col), * (dev_index_RC);
    bool* dev_encontrado;
    int size = numFilas * numColumnas;
    bool h_encontrado = true;
    int* h_index = { 0 };
    int* h_index_col = { 0 };
    int* h_index_fila = { 0 };
    int* h_index_RC = { 0 };

    int pos_encontrar = coordX * numColumnas + coordY;   //Posicion a ENCONTRAR en el vector 1D
    int color = h_tablero[pos_encontrar];
    int semilla = generarSemilla();
    //Reservar espacio en memoria para GPU (2 matrices y matriz resultado)
    hipMalloc((void**)&dev_Tablero, size * sizeof(int));
    hipMalloc((void**)&dev_index, sizeof(int));
    hipMalloc((void**)&dev_index_col, sizeof(int));
    hipMalloc((void**)&dev_index_fila, sizeof(int));
    hipMalloc((void**)&dev_index_RC, sizeof(int));
    hipMalloc(&dev_encontrado, sizeof(bool));

    //Copiamos datos a la GPU 
    hipMemcpy(dev_Tablero, h_tablero, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index, h_index, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_col, h_index_col, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_fila, h_index_fila, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_RC, h_index_RC, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_encontrado, &h_encontrado, sizeof(bool), hipMemcpyHostToDevice);

    dim3 dimGrid(gridX, gridY);
    dim3 dimBlock(hilosBloqueX, hilosBloqueY);

    //Segun si es alguno de los bloques especiales o es una jugada normal (66 --> B, 84 --> T,)
    int contenido = h_tablero[pos_encontrar];

    if (contenido == 'B')
    {
        kernelBomba << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }

    else if (contenido == 'T')
    {
        kernelTNT << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }

    else if (7 <= contenido && contenido <= 13) //Si es RC
    {
        int colorBorrar = contenido % 7;
        kernelRompeCabezas << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, colorBorrar, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }
    else //Si es bloque simple
    {
        int cont = 0;
        //Desde posici�n idicada se encuentran todos los caminos con el mismo color
        while (cont < numColumnas * numFilas)
        {
            while (h_encontrado)
            {

                kernelEncontrarCaminos << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, dev_index, pos_encontrar, dev_encontrado, color);
                hipMemcpy(&h_encontrado, dev_encontrado, sizeof(bool), hipMemcpyDeviceToHost);
                hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&h_index, dev_index, sizeof(int), hipMemcpyDeviceToHost);
            }

            if (h_tablero[cont] == -1)
            {
                pos_encontrar = cont;
                h_encontrado = 1;
            }
            cont += 1;
        }

        if ((int)h_index == 0 && vida >= 1)
        {
            vida = vida - 1;
        }
        h_index_fila = { 0 };
        h_index_col = { 0 };
        kernelEncontrarBomba << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar, dev_index_fila, dev_index_col);
        hipMemcpy(&h_index_fila, dev_index_fila, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_index_col, dev_index_col, sizeof(int), hipMemcpyDeviceToHost);
        dev_index_fila = 0;
        dev_index_col = 0;

        kernelEncontrarRompecabezasTNT << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar, dev_index_RC, semilla, dificultad);
        hipMemcpy(&h_index_RC, dev_index_RC, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
        dev_index_RC = 0;

    }
    h_index = { 0 };
    int iteraciones = 10;
    //Bucle para reemplazar las posiciones eliminadas mientras que se pueda hacer algun cambio y si no termine
    while (iteraciones > 0)
    {
        semilla = generarSemilla();
        kernelReemplazarPosiciones << <dimGrid, dimBlock >> > (dev_Tablero, numFilas, numColumnas, semilla, dificultad, dev_index);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_index, dev_index, sizeof(int), hipMemcpyDeviceToHost);
        iteraciones = (int)h_index;
    }
    hipFree(dev_encontrado);
    hipFree(dev_Tablero);
    hipFree(dev_index);
    hipFree(dev_index_fila);
    hipFree(dev_index_col);
    hipFree(dev_index_RC);

    return vida;
}

void main(int argc, char* argv[])
{
    //Declaracion variables
    int* h_tablero;
    int numFilas = 0;
    int numColumnas = 0;
    int coordenadaX;
    int coordenadaY;
    int size = 0;
    int dificultad = 0;
    char modoJuego = 'A';
    int vida = 5;

    //Saca las caracteristicas de nuestra tarjeta grafica
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    char* nombre = deviceProp.name;
    int maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;
    int maxThreadsSM = deviceProp.maxThreadsPerMultiProcessor;
    int maxBlockx = deviceProp.maxGridSize[0];
    int maxBlocky = deviceProp.maxGridSize[1];
    int maxGridX = deviceProp.maxThreadsDim[0];
    int maxGridY = deviceProp.maxThreadsDim[1];

    if (argc == 1)  //No se ha ejecutado por comando
    {
        printf("\nElija el modo de juego: A (Automatico) - M (Manual):  \n");
        scanf("%c", &modoJuego);
        printf("Modo de juego seleccionado: %c \n", modoJuego);
        printf("\nIntroduzca el numero de filas que tendra el tablero:  \n");
        scanf("%d", &numFilas);
        printf("\nIntroduzca el numero de columnas que tendra el tablero:  \n");
        scanf("%d", &numColumnas);
        printf("\nIntroduzca la dificultad del juego:  \n");
        scanf("%d", &dificultad);
    }

    else if (argc == -1)
    {
        printf("ERROR en ejecucion\n");
    }

    //Controla que tengamos los argumentso encesarios (tipo de ejecucion, dificultad, filas, columnas)
    else if (argc < 5)
    {
        printf("ERROR: faltan argumentos de entrada\n");
        printf("%d \n", argc);
    }
    //Controlan que no se pasen mas argumentos de los deseados
    else if (argc > 5)
    {
        printf("ERROR: sobran argumentos de entrada\n");
    }
    else //Realiza llamada con los 4 argumemtos
    {
        //Guarda los argumentos pasadas en las respectivas variables
        modoJuego = (char)argv[1];
        dificultad = std::stoi(argv[2]);    //Guarda valor argumentos usando funcion stoi para convertirlo a int
        numFilas = std::stoi(argv[3]);
        numColumnas = std::stoi(argv[4]);
    }

    size = numFilas * numColumnas;

    //Calcula el numero de hilos y bloques mas optimo para el tama�o de la matriz dado
    int hilosBloqueX = ceil(numColumnas / (float)2);
    int hilosBloqueY = ceil(numFilas / (float)2);
    int gridX = ceil(numColumnas / (float)hilosBloqueX);
    int gridY = ceil(numFilas / (float)hilosBloqueY);

    //Reservamos memoria para el tablero, ya que no esta inicializado
    h_tablero = (int*)malloc(size * sizeof(int));

    //Llamamos a la funcion que inicializa con valores aleatorios el tablero
    h_tablero = inicializarTablero(h_tablero, size, numColumnas, numFilas, dificultad, hilosBloqueX, hilosBloqueY, gridX, gridY);
    mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);

    //Se comprueba que se cumplan los limites de la tarjeta grafica
    if (hilosBloqueX >= maxBlockx && hilosBloqueY >= maxBlocky && gridX > maxGridX && gridY > maxGridY)
    {
        printf("\nSe sobrepasan las dimensiones asociadas a la tarjeta grafica  \n");
    }
    else
    {
        while (vida > 0)
        {
            if (modoJuego == 'M' || modoJuego == 'm')
            {
                printf("\nIntroduzca las coordenadas del bloque que desea eliminar (x, y):  \n");
                scanf("%d %d", &coordenadaX, &coordenadaY);
            }
            else if (modoJuego == 'A' || modoJuego == 'a' && numFilas > 0 && numColumnas > 0)
            {
                coordenadaX = (rand() % numFilas);
                coordenadaY = (rand() % numColumnas);

            }

            if ((coordenadaX < numFilas) && (coordenadaY < numColumnas) && (coordenadaX >= 0) && (coordenadaY >= 0))
            {
                vida = jugar(h_tablero, numFilas, numColumnas, coordenadaX, coordenadaY, dificultad, vida, hilosBloqueX, hilosBloqueY, gridX, gridY);
                mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
                printf("\nVida restante: %d \n", vida);
            }
            else
            {
                printf("\nLas coordenadas introducidas se encuentran fuera del rango del tablero [%d][%d] \n", numFilas, numColumnas);
            }
        }
        printf("\nPERDEDOR \n");
    }

}

