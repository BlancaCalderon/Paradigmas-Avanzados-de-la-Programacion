#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <string>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>


int vida = 5;

//Funcion que muestra el tablero por consola
void mostrarTablero(int* tablero, int numFilas, int numColumnas, int dificultad)
{
    int N = numFilas;
    int M = numColumnas;

    if (numFilas > numColumnas || numColumnas > numFilas) {
        N = numColumnas;
        M = numFilas;
    }

    for (int i = 0; i < numFilas; i++)
    {
        for (int j = 0; j < numColumnas; j++)
        {
            // printf("%d  ", tablero[i * N + j]);

            int num = tablero[i * N + j];
            if (num > dificultad)
            {
                if (7 <= num && num <= 13)
                {
                    printf("RC%d  ", num % 7);
                }
                else
                {
                    printf("%c  ", (char)num);
                }
            }
            else
            {
                printf("%d  ", num);
            }

        }
        printf("\n");
    }
    printf("\n");
}

//Kernel que elimina la fila y columna de la posicion pasada (Bomba)
__global__ void kernelBomba(int* dev_tablero, int numFila, int numCol, int pos_encontrar)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;        //Posicion en la que nos encontramos

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna a borrar
    int filaBorrar = pos_encontrar / numCol;
    int colBorrar = pos_encontrar - filaBorrar * numCol;

    //si posici�n actual esta en la fila o columna que queremos borrar
    if (filaBorrar == filaActual || colBorrar == colActual && 0 <= filaActual <= numFila && 0 <= colActual <= numCol)
    {
        dev_tablero[pos] = -1; //Indicamos que se borra
    }
    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial
}


//Kernel que elimina los elementos adyacentes a una posici�n (radio 4 elementos) (TNT)
__global__ void kernelTNT(int* dev_tablero, int numFila, int numCol, int pos_encontrar)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;        //Posicion en la que nos encontramos

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna a borrar teniendo en cuenta el rango
    int filaBorrar = pos_encontrar / numCol;
    int colBorrar = pos_encontrar - filaBorrar * numCol;

    int filaBorrarDer = filaBorrar + 4;
    int colBorrarAbajo = colBorrar + 4;
    int filaBorrarIzq = filaBorrar - 4;
    int colBorrarArriba = colBorrar - 4;

    //si posici�n actual es adyacente y esta dentro del rango que queremos borrar (4)
    if (filaBorrarIzq <= filaActual <= filaBorrarDer && colBorrarArriba <= colActual <= colBorrarAbajo && 0 <= filaActual <= numFila && 0 <= colActual <= numCol)
    {
        dev_tablero[pos] = -1; //Indicamos que se borra
    }
    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial
}


//Kernel que elimina todos las posiciones del color indicado (ROMPECABEZAS)
__global__ void kernelRompeCabezas(int* dev_tablero, int numFila, int numCol, int color, int pos_encontrar)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;        //Posicion en la que nos encontramos

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //si posici�n actual tiene el color indicado se elimina
    if (dev_tablero[pos] == color)
    {
        dev_tablero[pos] = -1; //Indicamos que se borra
    }
    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    dev_tablero[pos_encontrar] = -1;              //Eliminamos bloque especial
}


//Kernel que lleva a cabo la generacion del tablero de forma aleatoria
__global__ void kernelGenerarTablero(int* dev_tablero, int dev_semilla, int dificultad)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t state;
    hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
    int color = abs((int)(hiprand(&state) % dificultad) + 1);
    dev_tablero[pos] = color; //Rellena tablero con numeros aleatorios entre 1 y 6
}

__global__ void kernelReemplazarPosiciones(int* dev_tablero, int numFila, int numCol, int dev_semilla, int dificultad, int* dev_index)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;        //Posicion en la que nos encontramos
    dev_index[0] = 0;                                       //Lo utilizamos para contabilizar el numero de llamadas que hay que realizar al kernel 

    if (dev_tablero[pos] == -1)
    {

        int filaActual = pos / numCol;
        int colActual = pos - filaActual * numCol;
        if (filaActual > 0 && filaActual <= numFila && dev_tablero[pos - numCol] != -1)
        {
            __syncthreads();

            dev_tablero[pos] = dev_tablero[pos - numCol];
            dev_tablero[pos - numCol] = -1;
            atomicAdd(&dev_index[0], 1);
            __syncthreads();
        }
        else if (dev_tablero[pos - numCol] != -1)
        {
            hiprandState_t state;
            hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
            int color = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
            dev_tablero[pos] = color;
            atomicAdd(&dev_index[0], 1);
            __syncthreads();
        }
    }

}

__global__ void kernelEncontrarCaminos(int* dev_tablero, int numFila, int numCol, int* dev_index, int pos_encontrar, bool* dev_encontrado, int color)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;        //Posicion en la que nos encontramos
    bool encontrado = false;
    bool camino_invalido = false;
    int posAux;
    int index = 0;

    //Recorrer 1� fila y 2�Columna del tablero en la que se encuentra la celda de POS
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;
    int ultima_posicion = pos;

    if ((dev_tablero[pos] == color || dev_tablero[pos] == -1) && pos_encontrar == pos)
    {
        encontrado = false;
        posAux = pos;

        while ((posAux < numCol * numFila) && !encontrado && !camino_invalido)
        {
            int sigfila = (posAux + 1) / numCol;                 //Fila en la que se encuentra el siguiente elemento
            int sigcol = (posAux + 1) - sigfila * numCol;       //Columna en la que se encuentra el siguiente elemento

            int fila_anterior = (posAux - 1) / numCol;                 //Fila en la que se encuentra el elemento ANTERIOR
            int col_anterior = (posAux - 1) - fila_anterior * numCol; //Columna en la que se encuentra el elemento anterior

            int posSigFila = (posAux + numCol) / numCol;
            int fila_actual = posAux / numCol;
            int col_actual = posAux - fila_actual * numCol;

            if (color == dev_tablero[posAux + 1] && sigcol > 0 && (posAux + 1) != ultima_posicion)          //Nos desplazamos a la derecha
            {
                index += 1;
                ultima_posicion = posAux;
                posAux += 1;
                dev_tablero[posAux] = -1;
            }
            else if (color == dev_tablero[posAux + numCol] && (posAux + numCol) != ultima_posicion && (posAux + numCol) < numCol * numFila)  //Hacia abajo
            {
                ultima_posicion = posAux;
                posAux = posAux + numCol;
                index += 1;

                dev_tablero[posAux] = -1;
            }
            else if (color == dev_tablero[posAux - 1] && col_anterior > 0 && (posAux - 1) != ultima_posicion)           //Izquierda
            {
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux - 1;
                dev_tablero[posAux] = -1;
            }
            else if (color == dev_tablero[posAux - numCol] && (posAux - numCol) != ultima_posicion && (posAux - numCol) >= 0 && filaActual > 0 && filaActual < numFila)  //ARRIBA
            {
                index += 1;
                ultima_posicion = posAux;
                posAux = posAux - numCol;

                dev_tablero[posAux] = -1;
            }
            else
            {
                if (index > 0) {
                    atomicAdd(&dev_index[0], 1);
                    encontrado = true;
                }
                else
                {
                    encontrado = false;
                }
                __syncthreads();
                camino_invalido = true;
            }

        }
        dev_encontrado[0] = encontrado;

    }
    dev_encontrado[0] = encontrado;

    __syncthreads(); //Esperamos a que todos los hilos del mismo bloque hayan ejecutado el if antes de establecer la posicion a encontrar en -1
    if (dev_index[0] >= 1)
    {
        dev_tablero[pos_encontrar] = -1;              //Establecemos la posicion a encontrar en -1
    }



}

__global__ void kernelEncontrarBomba(int* dev_tablero, int numFila, int numCol, int pos_encontrar, int* dev_index_fila, int* dev_index_col)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna de la posici�n a encontrar
    int filaEncontrar = pos_encontrar / numCol;
    int colEncontrar = pos_encontrar - filaEncontrar * numCol;

    if (filaActual == filaEncontrar && (int)dev_index_fila > 5 && dev_tablero[pos] == -1)
    {
        atomicAdd(&dev_index_fila[0], 1);
    }

    if (colActual == colEncontrar && (int)dev_index_col > 5 && dev_tablero[pos] == -1)
    {
        atomicAdd(&dev_index_col[0], 1);
    }

    __syncthreads();
    if (dev_index_fila[0] != dev_index_col[0])
    {
        if ((dev_index_fila[0] == 5 && dev_index_col[0] == 1) || (dev_index_col[0] == 5 && dev_index_fila[0] == 1))
        {
            dev_tablero[pos_encontrar] = 'B';
        }
    }
    dev_index_fila[0] = 0;
    dev_index_col[0] = 0;
}

__global__ void kernelEncontrarRompecabezasTNT(int* dev_tablero, int numFila, int numCol, int pos_encontrar, int* dev_index, int dev_semilla, int dificultad)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    //Calcula fila y columna de la posici�n actual
    int filaActual = pos / numCol;
    int colActual = pos - filaActual * numCol;

    //Calcula fila y columna de la posici�n a encontrar
    int filaEncontrar = pos_encontrar / numCol;
    int colEncontrar = pos_encontrar - filaEncontrar * numCol;

    if (dev_tablero[pos] == -1)
    {
        atomicAdd(&dev_index[0], 1);
    }
    __syncthreads();

    if (dev_index[0] == 6 && pos == pos_encontrar)
    {
        dev_tablero[pos_encontrar] = 'T';
    }
    else if (dev_index[0] >= 7 && pos == pos_encontrar)
    {
        hiprandState_t state;
        hiprand_init(dev_semilla, pos, 0, &state); //hiprand_init(semilla, secuencia, offset, estado) secuencia dgenera diferentes secuencias de numeros aleatorio a partir de la misma semilla y offset genera numeros aleatorio s a partir de una secuencia y una semilla  CurandState hiprandState;
        int color = abs((int)(hiprand(&state) % dificultad) + 1);  //Rellena tablero con numeros aleatorios entre 1 y 6
        int colorS = 7 + color;
        dev_tablero[pos_encontrar] = colorS;
    }
    dev_index[0];

}


//Inicializamos el tablero
int* inicializarTablero(int* h_tablero, int size, int dificultad)
{
    int* (dev_Tablero);

    //Reservar espacio en memoria para GPU (2 matrices y matriz resultado)
    hipMalloc((void**)&dev_Tablero, size * sizeof(int));

    //Copiamos datos a la GPU 
    hipMemcpy(dev_Tablero, h_tablero, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsInBlock(size);

    unsigned int semilla = time(NULL);
    kernelGenerarTablero << <1, threadsInBlock >> > (dev_Tablero, semilla, dificultad);

    // Copiamos de la GPU a la CPU
    hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);

    return h_tablero;

}

//Funcion que llama a kernel para encontrar todos los caminos hacia bloque indicado
int encontrarCamino(int* h_tablero_original, int numFilas, int numColumnas, int coordX, int coordY, int dificultad, int vida)
{
    int* h_tablero = h_tablero_original;
    int* (dev_Tablero), * (dev_index), * (dev_index_fila), * (dev_index_col), * (dev_index_RC), * (dev_index_T);
    bool* dev_encontrado;
    int size = numFilas * numColumnas;
    bool h_encontrado = true;
    int* h_index = { 0 };
    int* h_index_col = { 0 };
    int* h_index_fila = { 0 };
    int* h_index_RC = { 0 };
    int pos_encontrar = coordX * numFilas + coordY;   //Posicion a ENCONTRAR en el vector 1D
    unsigned int semilla = time(NULL);

    if (numColumnas > numFilas) //Si matriz asimetrica con mas columnas que filas
    {
        pos_encontrar = coordX * numColumnas + coordY;   //Posicion a ENCONTRAR en el vector 1D   
    }

    int color = h_tablero[pos_encontrar];

    //Reservar espacio en memoria para GPU (2 matrices y matriz resultado)
    hipMalloc((void**)&dev_Tablero, size * sizeof(int));
    hipMalloc((void**)&dev_index, sizeof(int));
    hipMalloc((void**)&dev_index_col, sizeof(int));
    hipMalloc((void**)&dev_index_fila, sizeof(int));
    hipMalloc((void**)&dev_index_RC, sizeof(int));
    hipMalloc(&dev_encontrado, sizeof(bool));

    //Copiamos datos a la GPU 
    hipMemcpy(dev_Tablero, h_tablero, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index, h_index, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_col, h_index_col, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_fila, h_index_fila, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_index_RC, h_index_RC, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_encontrado, &h_encontrado, sizeof(bool), hipMemcpyHostToDevice);

    dim3 threadsInBlock(size);

    //Segun si es alguno de los bloques especiales o es una jugada normal (66 --> B, 84 --> T,)
    int contenido = h_tablero[pos_encontrar];

    if (contenido == 'B')
    {
        kernelBomba << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }

    else if (contenido == 'T')
    {
        kernelTNT << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }

    else if (7 <= contenido && contenido <= 13) //Si es RC
    {
        int colorBorrar = contenido % 7;
        kernelRompeCabezas << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, colorBorrar, pos_encontrar);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
    }
    else //Si es bloque simple
    {
        int cont = 0;
        //Desde posici�n idicada se encuentran todos los caminos con el mismo color
        while (cont < numColumnas * numFilas)
        {
            while (h_encontrado)
            {

                kernelEncontrarCaminos << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, dev_index, pos_encontrar, dev_encontrado, color);
                hipMemcpy(&h_encontrado, dev_encontrado, sizeof(bool), hipMemcpyDeviceToHost);
                hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&h_index, dev_index, sizeof(int), hipMemcpyDeviceToHost);;
            }

            if (h_tablero[cont] == -1)
            {
                pos_encontrar = cont;
                h_encontrado = 1;
            }
            cont += 1;
        }
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);

        if ((int)h_index == 0 && vida >= 1)
        {
            vida = vida - 1;
        }
        h_index_fila = { 0 };
        h_index_col = { 0 };
        mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
        kernelEncontrarBomba << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar, dev_index_fila, dev_index_col);
        hipMemcpy(&h_index_fila, dev_index_fila, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_index_col, dev_index_col, sizeof(int), hipMemcpyDeviceToHost);


        kernelEncontrarRompecabezasTNT << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, pos_encontrar, dev_index_RC, semilla, dificultad);
        hipMemcpy(&h_index_RC, dev_index_RC, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);

    }


    h_index = { 0 };
    int iteraciones = 10;
    //Bucle para reemplazar las posiciones eliminadas mientras que se pueda hacer algun cambio y si no termine
    while (iteraciones > 0)
    {
        kernelReemplazarPosiciones << <1, threadsInBlock >> > (dev_Tablero, numFilas, numColumnas, semilla, dificultad, dev_index);
        hipMemcpy(h_tablero, dev_Tablero, size * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_index, dev_index, sizeof(int), hipMemcpyDeviceToHost);
        iteraciones = (int)h_index;
    }

    hipFree(dev_encontrado);
    hipFree(dev_Tablero);
    hipFree(dev_index);
    hipFree(dev_index_fila);
    hipFree(dev_index_col);
    hipFree(dev_index_RC);

    return vida;
}

void main(int argc, char* argv[])
{
    //Declaracion variables
    int* h_tablero;
    int numFilas = 0;
    int numColumnas = 0;
    int coordenadaX;
    int coordenadaY;
    int size = 0;
    int dificultad = 4;
    bool terminado = false;
    char modoJuego = 'A';

    //int h_tablero[27] = { 3,3,3,3,3,3,3,4,4,4,4,3,1,3,3,3,4,3,3,3,4,3,3,4,3,4,4 };
    //int h_tablero[27] = { 3,3,3,3,4,3,3,4,4,1,4,4,1,3,1,3,1,3,3,3,4,1,1,4,3 };
    //  int h_tablero[27] = { 3,3,3,3,3,3,4,4,4,4,4,4,4,4,4,4,4,4,4,4,4,1,1,4,4,4,4 };
    //  mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
      //int h_tablero[25] = { 3,2,1,5,5,3,3,6,7,3,9,3,'B',3,1,3,1,3,3,3,4,1,1,4,3 };

    if (argc == 1)  //No se ha ejecutado por comando
    {
        printf("\nElija el modo de juego: A (Automatico) - M (Manual):  \n");
        scanf("%c", &modoJuego);
        printf("Modo de juego seleccionado: %c \n", modoJuego);
        printf("\nIntroduzca el numero de filas que tendra el tablero:  \n");
        scanf("%d", &numFilas);
        printf("\nIntroduzca el numero de columnas que tendra el tablero:  \n");
        scanf("%d", &numColumnas);
        printf("\nIntroduzca la dificultad del juego:  \n");
        scanf("%d", &dificultad);
    }

    else if (argc == -1)
    {
        printf("ERROR en ejecucion\n");
    }
    //Controla que tengamos los argumentso encesarios (tipo de ejecucion, dificultad, filas, columnas)
    else if (argc < 5)
    {
        printf("ERROR: faltan argumentos de entrada\n");
        printf("%d \n", argc);
    }
    //Controlan que no se pasen mas argumentos de los deseados
    else if (argc > 5)
    {
        printf("ERROR: sobran argumentos de entrada\n");
    }
    else //Realiza llamada con los 4 argumemtos
    {
        //Guarda los argumentos pasadas en las respectivas variables
        modoJuego = (char)argv[1];
        dificultad = std::stoi(argv[2]);    //Guarda valor argumentos usando funcion stoi para convertirlo a int
        numFilas = std::stoi(argv[3]);
        numColumnas = std::stoi(argv[4]);
    }

    size = numColumnas * numFilas;

    //Reservamos memoria para el tablero, ya que no esta inicializado
    h_tablero = (int*)malloc(size * sizeof(int));

    //Llamamos a la funcion que inicializa con valores aleatorios el tablero
    h_tablero = inicializarTablero(h_tablero, size, dificultad);

    mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);

    while (vida > 0)
    {
        if (modoJuego == 'M' || modoJuego == 'm')
        {
            printf("\nIntroduzca las coordenadas del bloque que desea eliminar (x, y):  \n");
            scanf("%d %d", &coordenadaX, &coordenadaY);
        }
        else if (modoJuego == 'A' || modoJuego == 'a')
        {
            coordenadaX = (rand() % numFilas);
            coordenadaY = (rand() % numColumnas);
            printf("\nCoordenadas (%d, %d)  \n", coordenadaX, coordenadaY);
        }
        if ((coordenadaX < numFilas) && (coordenadaY < numColumnas) && (coordenadaX >= 0) && (coordenadaY >= 0))
        {
            vida = encontrarCamino(h_tablero, numFilas, numColumnas, coordenadaX, coordenadaY, dificultad, vida);
            printf("\nVida restante: %d \n", vida);
            mostrarTablero(h_tablero, numFilas, numColumnas, dificultad);
        }
        else
        {
            printf("\nLas coordenadas introducidas se encuentran fuera del rango del tablero [%d][%d] \n", numFilas, numColumnas);
        }


    }
    printf("\nPERDEDOR \n");
}



